#include "hip/hip_runtime.h"
/*******************************************************************************
 * This file is part of mdcore.
 * Coypright (c) 2012 Pedro Gonnet (pedro.gonnet@durham.ac.uk)
 * Copyright (c) 2022 T.J. Sego
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published
 * by the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU Lesser General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 * 
 ******************************************************************************/

// TODO: implement hook for potentials by particles

/* Include configuratin header */
#include <mdcore_config.h>

/* Include some standard header files */
#include <stdlib.h>
#include <stdio.h>
#include <pthread.h>
#include <math.h>
#include <float.h>
#include <string.h>
#include <limits.h>

/* Include headers for overloaded vector functions. */
#include "cutil_math.h"

#include <hip/hip_runtime.h>

/* Include some conditional headers. */
#ifdef HAVE_MPI
    #include <mpi.h>
#endif

#include <tfTaskScheduler.h>

/* Force single precision. */
#ifndef FPTYPE_SINGLE
    #define FPTYPE_SINGLE 1
#endif

/* Disable vectorization for the nvcc compiler's sake. */
#undef __SSE__
#undef __SSE2__
#undef __ALTIVEC__
#undef __AVX__

/* Include local headers */
#include <cycle.h>
#include <tf_errs.h>
#include <tf_fptype.h>
#include <tf_lock.h>
#include "tfParticle_cuda.h"
#include <tfSpace_cell.h>
#include <tfSpace.h>
#include <tfTask.h>
#include "tfPotential_cuda.h"
#include "tfBoundaryConditions_cuda.h"
#include <tfFlux.h>
#include <tfEngine.h>
#include "tfRunner_cuda.h"
#include "tfFlux_cuda.h"

#ifndef CPU_TPS
#include <ctime>
#define CPU_TPS CLOCKS_PER_SEC
#endif


using namespace TissueForge;


/* the error macro. */
#define error(id)				(engine_err = errs_register(id, engine_err_msg[-(id)], __LINE__, __FUNCTION__, __FILE__))
#define cuda_error(id)			(engine_err = errs_register(id, hipGetErrorString(hipGetLastError()), __LINE__, __FUNCTION__, __FILE__))
#define cuda_safe_call(f)       { if(f != hipSuccess) return cuda_error(engine_err_cuda); }
#define cuda_safe_call_e(f, _ret_ok)       { if(f != _ret_ok) return cuda_error(engine_err_cuda); }

/* The parts (non-texture access). */
__constant__ cuda::Particle *cuda_parts;
__constant__ float *cuda_part_states;
static unsigned int engine_cuda_nr_states = 0;

// Boundary conditions
__constant__ cuda::BoundaryConditions cuda_bcs;
static cuda::BoundaryConditions cuda_bcs_dev;

/* Diagonal entries and potential index lookup table. */
__constant__ int *cuda_pind;
__constant__ int *cuda_pind_cluster;
__constant__ int *cuda_pind_bcs[6];

/* The mutex for accessing the cell pair list. */
__device__ int cuda_cell_mutex = 0;
__device__ int cuda_barrier = 0;
__device__ volatile int NAMD_barrier = 0;

/* The index of the next free cell pair. */
__device__ int cuda_pair_next = 0;

/* The list of cell pairs. */

__device__ int *cuda_taboo;
#ifdef TASK_TIMERS
/*x = block y = type z = start w = end*/
__device__ int4 NAMD_timers[26*10000];
#endif

/* The list of tasks. */
__constant__ struct cuda::task_cuda *cuda_tasks;
__constant__ int cuda_nr_tasks = 0;

/* The per-SM task queues. */
__device__ struct cuda::queue_cuda cuda_queues[ cuda_maxqueues ];
__device__ struct cuda::queue_cuda cuda_sorts[ cuda_maxqueues ];
__constant__ int cuda_queue_size;

/* Some constants. */
__constant__ float cuda_dt = 0.0f;
__constant__ float cuda_cutoff2 = 0.0f;
__constant__ float cuda_cutoff = 0.0f;
__constant__ float cuda_dscale = 0.0f;
__constant__ float cuda_maxdist = 0.0f;
__constant__ int cuda_maxtype = 0;

/* Sortlists for the Verlet algorithm. */
__device__ unsigned int *cuda_sortlists = NULL;

/* Cell origins. */
__constant__ float *cuda_corig;

// Cell dimensions
__constant__ float3 *cuda_cdims;

// Cell flags
__constant__ unsigned int *cuda_cflags;

/* Potential energy. */
__device__ float cuda_epot = 0.0f, cuda_epot_out;

/* Timers. */
__device__ float cuda_timers[ tid_count ];

// Fluxes
extern __constant__ int *cuda_fxind;
extern __constant__ struct cuda::Fluxes *cuda_fluxes;

// Potential data

#define ENGINE_CUDA_PIND_WIDTH      3
#define ENGINE_CUDA_POT_WIDTH_ALPHA 3
#define ENGINE_CUDA_POT_WIDTH_DATAF 6
#define ENGINE_CUDA_POT_WIDTH_DATAI 2
#define ENGINE_CUDA_DPD_WIDTH_CF    3
#define ENGINE_CUDA_DPD_WIDTH_DATAF 2
#define ENGINE_CUDA_DPD_WIDTH_DATAI 1

static int *pind_bcs_cuda[engine_maxgpu][6];

static hipArray *cuda_pot_alpha[engine_maxgpu], *cuda_pot_cluster_alpha[engine_maxgpu], *cuda_pot_bcs_alpha[engine_maxgpu][6];
static hipArray *cuda_pot_c[engine_maxgpu], *cuda_pot_cluster_c[engine_maxgpu], *cuda_pot_bcs_c[engine_maxgpu][6];
static hipArray *cuda_pot_dataf[engine_maxgpu], *cuda_pot_cluster_dataf[engine_maxgpu], *cuda_pot_bcs_dataf[engine_maxgpu][6];
static hipArray *cuda_pot_datai[engine_maxgpu], *cuda_pot_cluster_datai[engine_maxgpu], *cuda_pot_bcs_datai[engine_maxgpu][6];

static hipTextureObject_t tex_pot_alpha[engine_maxgpu], tex_pot_cluster_alpha[engine_maxgpu], tex_pot_bcs_alpha[engine_maxgpu][6];
static hipTextureObject_t tex_pot_c[engine_maxgpu], tex_pot_cluster_c[engine_maxgpu], tex_pot_bcs_c[engine_maxgpu][6];
static hipTextureObject_t tex_pot_dataf[engine_maxgpu], tex_pot_cluster_dataf[engine_maxgpu], tex_pot_bcs_dataf[engine_maxgpu][6];
static hipTextureObject_t tex_pot_datai[engine_maxgpu], tex_pot_cluster_datai[engine_maxgpu], tex_pot_bcs_datai[engine_maxgpu][6];

__constant__ hipTextureObject_t cuda_tex_pot_alpha, cuda_tex_pot_cluster_alpha, cuda_tex_pot_bcs_alpha[6];
__constant__ hipTextureObject_t cuda_tex_pot_c, cuda_tex_pot_cluster_c, cuda_tex_pot_bcs_c[6];
__constant__ hipTextureObject_t cuda_tex_pot_dataf, cuda_tex_pot_cluster_dataf, cuda_tex_pot_bcs_dataf[6];
__constant__ hipTextureObject_t cuda_tex_pot_datai, cuda_tex_pot_cluster_datai, cuda_tex_pot_bcs_datai[6];

static hipArray *cuda_dpd_cfs[engine_maxgpu], *cuda_dpd_cluster_cfs[engine_maxgpu], *cuda_dpd_bcs_cfs[engine_maxgpu][6];
static hipArray *cuda_dpd_dataf[engine_maxgpu], *cuda_dpd_cluster_dataf[engine_maxgpu], *cuda_dpd_bcs_dataf[engine_maxgpu][6];
static hipArray *cuda_dpd_datai[engine_maxgpu], *cuda_dpd_cluster_datai[engine_maxgpu], *cuda_dpd_bcs_datai[engine_maxgpu][6];

static hipTextureObject_t tex_dpd_cfs[engine_maxgpu], tex_dpd_cluster_cfs[engine_maxgpu], tex_dpd_bcs_cfs[engine_maxgpu][6];
static hipTextureObject_t tex_dpd_dataf[engine_maxgpu], tex_dpd_cluster_dataf[engine_maxgpu], tex_dpd_bcs_dataf[engine_maxgpu][6];
static hipTextureObject_t tex_dpd_datai[engine_maxgpu], tex_dpd_cluster_datai[engine_maxgpu], tex_dpd_bcs_datai[engine_maxgpu][6];

__constant__ hipTextureObject_t cuda_tex_dpd_cfs, cuda_tex_dpd_cluster_cfs, cuda_tex_dpd_bcs_cfs[6];
__constant__ hipTextureObject_t cuda_tex_dpd_dataf, cuda_tex_dpd_cluster_dataf, cuda_tex_dpd_bcs_dataf[6];
__constant__ hipTextureObject_t cuda_tex_dpd_datai, cuda_tex_dpd_cluster_datai, cuda_tex_dpd_bcs_datai[6];

__constant__ int cuda_pots_max, cuda_pots_cluster_max, cuda_pots_bcs_max[6], cuda_dpds_max, cuda_dpds_cluster_max, cuda_dpds_bcs_max[6];


/* Map sid to shift vectors. */
__constant__ float cuda_shiftn[13*3] = {
     5.773502691896258e-01,  5.773502691896258e-01,  5.773502691896258e-01,
     7.071067811865475e-01,  7.071067811865475e-01,  0.0                  ,
     5.773502691896258e-01,  5.773502691896258e-01, -5.773502691896258e-01,
     7.071067811865475e-01,  0.0                  ,  7.071067811865475e-01,
     1.0                  ,  0.0                  ,  0.0                  ,
     7.071067811865475e-01,  0.0                  , -7.071067811865475e-01,
     5.773502691896258e-01, -5.773502691896258e-01,  5.773502691896258e-01,
     7.071067811865475e-01, -7.071067811865475e-01,  0.0                  ,
     5.773502691896258e-01, -5.773502691896258e-01, -5.773502691896258e-01,
     0.0                  ,  7.071067811865475e-01,  7.071067811865475e-01,
     0.0                  ,  1.0                  ,  0.0                  ,
     0.0                  ,  7.071067811865475e-01, -7.071067811865475e-01,
     0.0                  ,  0.0                  ,  1.0                  ,
     };
__constant__ float cuda_shift[13*3] = {
     1.0,  1.0,  1.0,
     1.0,  1.0,  0.0,
     1.0,  1.0, -1.0,
     1.0,  0.0,  1.0,
     1.0,  0.0,  0.0,
     1.0,  0.0, -1.0,
     1.0, -1.0,  1.0,
     1.0, -1.0,  0.0,
     1.0, -1.0, -1.0,
     0.0,  1.0,  1.0,
     0.0,  1.0,  0.0,
     0.0,  1.0, -1.0,
     0.0,  0.0,  1.0,
    };
    
/* The cell edge lengths and space dimensions. */
__constant__ float hip/hip_runtime.h[3];
__constant__ float cuda_dim[3];
    
    
/**
 * @brief Lock a device mutex.
 *
 * @param m The mutex.
 *
 * Loops until the mutex can be set. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ void cuda_mutex_lock(int *m) {
    TIMER_TIC
    while(atomicCAS(m, 0, 1) != 0);
    TIMER_TOC(tid_mutex)
}


/**
 * @brief Attempt to lock a device mutex.
 *
 * @param m The mutex.
 *
 * Try to grab the mutex. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ int cuda_mutex_trylock(int *m) {
    TIMER_TIC
    int res = atomicCAS(m, 0, 1) == 0;
    TIMER_TOC(tid_mutex)
    return res;
}


/**
 * @brief Lock a device mutex with an additional condition.
 *
 * @param m The mutex.
 * @param c the condition
 *
 * @return @c 1 if the mutex could be locked or zero if the condition @c c
 * was reached first.
 *
 * Loops until the mutex can be set or until @c *c is non-zero.
 * Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ int cuda_mutex_lock_cond(int *m, int *c) {
    TIMER_TIC
    while(atomicCAS(c, 0, 0) == 0)
        if(atomicCAS(m, 0, 1) == 0) {
            TIMER_TOC(tid_mutex)
            return 1;
            }
    TIMER_TOC(tid_mutex)
    return 0;
}


/**
 * @brief Unlock a device mutex.
 *
 * @param m The mutex.
 *
 * Does not check if the mutex had been locked.
 */

__device__ void cuda_mutex_unlock(int *m) {
    TIMER_TIC
    atomicExch(m, 0);
    TIMER_TOC(tid_mutex)
}
    
    
/**
 * @brief Get a task ID from the given queue.
 *
 */
 
__device__ int cuda_queue_gettask(struct cuda::queue_cuda *q) {

    int ind, tid = -1;
    
    /* Don't even try... */
    if(q->rec_count == q->count)
        return -1;

    /* Get the index of the next task. */
    ind = atomicAdd(&q->first, 1);
        
    /* Wrap the index. */
    ind %= cuda_queue_size; 

    /* Loop until there is a valid task at that index. */
    while(q->rec_count < q->count &&(tid = q->data[ind]) < 0);
    
    /* Scratch the task from the queue */
    if(tid >= 0)
        q->data[ind] = -1;

    /* Return the acquired task ID. */
    return tid;
    
}


/**
 * @brief Put a task onto the given queue.
 *
 * @param tid The task ID to add to the end of the queue.
 */
 
__device__ void cuda_queue_puttask(struct cuda::queue_cuda *q, int tid) {

    int ind;

    /* Get the index of the next task. */
    ind = atomicAdd(&q->last, 1) % cuda_queue_size;
    
    /* Wait for the slot in the queue to be empty. */
    while(q->data[ind] != -1);

    /* Write the task back to the queue. */
    q->data[ind] = tid;
    
}
    
    
/**
 * @brief Get the ID of the block's SM.
 */
 
__noinline__ __device__ uint get_smid(void) {
    uint ret;
    asm("mov.u32 %0, %smid;" : "=r"(ret));
    return ret;
}


/**
 * @brief Get a task from the given task queue.
 *
 * Picks tasks from the queue sequentially and checks if they
 * can be computed. If not, they are returned to the queue.
 *
 * This routine blocks until a valid task is picked up, or the
 * specified queue is empty.
 */
 
__device__ int runner_cuda_gettask(struct cuda::queue_cuda *q, int steal) {

    int tid = -1;
    int cid, cjd;
    
    TIMER_TIC
    
    /* Main loop. */
    while((tid = cuda_queue_gettask(q)) >= 0) {
    
        /* If this task is not even free, don't even bother. */
        if(!cuda_tasks[tid].wait) {
    
            /* Dfferent options for different tasks. */
            if(cuda_tasks[tid].type == task_type_sort) {
            
                /* No locking needed. */
                break;
            
            }
            else if(cuda_tasks[tid].type == task_type_self) {
            
                /* Decode this task. */
                cid = cuda_tasks[tid].i;

                /* Lock down this task? */
                if(cuda_mutex_trylock(&cuda_taboo[ cid ]))
                    break;
                        
            }
            else if(cuda_tasks[tid].type == task_type_pair) {
            
                /* Decode this task. */
                cid = cuda_tasks[tid].i;
                cjd = cuda_tasks[tid].j;

                /* Lock down this task? */
                if(cuda_mutex_trylock(&cuda_taboo[ cid ]))
                    if(cuda_mutex_trylock(&cuda_taboo[ cjd ])) 
                        break;
                    else
                        cuda_mutex_unlock(&cuda_taboo[ cid ]);
                        
            }

        }
                
        /* Put this task back into the queue. */
        cuda_queue_puttask(q, tid);
    
    }
        
    /* Put this task into the recycling queue, if needed. */
    if(tid >= 0) {
        if(steal)
            atomicSub((int *)&q->count, 1);
        else
            q->rec_data[ atomicAdd((int *)&q->rec_count, 1) ] = tid;
    }
        
    TIMER_TOC(tid_queue);
        
    /* Return whatever we got. */
    return tid;

}

__device__ int runner_cuda_gettask_nolock(struct cuda::queue_cuda *q, int steal) {

    int tid = -1/*,cid,cjd*/;
    
    TIMER_TIC
    
    /* Main loop. */
    while((tid = cuda_queue_gettask(q)) >= 0) {
    
        /* If this task is not even free, don't even bother. */
        if(!cuda_tasks[tid].wait) {
    
            break;

        }
        /*if(cuda_tasks[tid].type == task_type_pair)
        {
	       	cid = cuda_tasks[tid].i;
	       	cjd = cuda_tasks[tid].j;
	       	if(!(cuda_taboo[cid] || cuda_taboo[cjd]))
	       		break;
       	}else{
       		break;
       	}*/
       	
                
        /* Put this task back into the queue. */
        cuda_queue_puttask(q, tid);
    
    }
        
    /* Put this task into the recycling queue, if needed. */
    if(tid >= 0) {
        if(steal)
            atomicSub((int *)&q->count, 1);
        else
            q->rec_data[ atomicAdd((int *)&q->rec_count, 1) ] = tid;
    }
        
    TIMER_TOC(tid_queue);
        
    /* Return whatever we got. */
    return tid;

}

__device__ inline void w_cubic_spline_cuda(float r2, float h, float *result) {
    float r = rsqrt(r2);
    float x = r/h;
    float y;
    
    if(x < 1.f) {
        float x2 = x * x;
        y = 1.f - (3.f / 2.f) * x2 + (3.f / 4.f) * x2 * x;
    }
    else if(x >= 1.f && x < 2.f) {
        float arg = 2.f - x;
        y = (1.f / 4.f) * arg * arg * arg;
    }
    else {
        y = 0.f;
    }
    
    *result = y / (M_PI * h * h * h);
}

__device__ inline void w_cubic_spline_cuda(float r2, float h, float *result, float *_r) {
    float r = sqrt(r2);
    float x = 1.f/(r*h);
    float y;
    
    if(x < 1.f) {
        float x2 = x * x;
        y = 1.f - (3.f / 2.f) * x2 + (3.f / 4.f) * x2 * x;
    }
    else if(x >= 1.f && x < 2.f) {
        float arg = 2.f - x;
        y = (1.f / 4.f) * arg * arg * arg;
    }
    else {
        y = 0.f;
    }
    
    *result = y / (M_PI * h * h * h);
    *_r = r;
}

__device__ inline void w_cubic_spline_cuda_nr(float r2, float h, float *result) {
    float r = sqrt(r2);
    float x = 1.f/(r*h);
    float y;
    
    if(x < 1.f) {
        float x2 = x * x;
        y = 1.f - (3.f / 2.f) * x2 + (3.f / 4.f) * x2 * x;
    }
    else if(x >= 1.f && x < 2.f) {
        float arg = 2.f - x;
        y = (1.f / 4.f) * arg * arg * arg;
    }
    else {
        y = 0.f;
    }
    
    *result = y / (M_PI * h * h * h);
}
    
/**
 * @brief Copy bulk memory in a strided way.
 *
 * @param dest Pointer to destination memory.
 * @param source Pointer to source memory.
 * @param count Number of bytes to copy, must be a multiple of sizeof(int).
 */
 
__device__ inline void cuda_memcpy(void *dest, void *source, int count) {

    int k;
    int *idest = (int *)dest, *isource = (int *)source;

    int threadID = threadIdx.x;
    
    TIMER_TIC
    
    /* Copy the data in chunks of sizeof(int). */
    for(k = threadID ; k < count/sizeof(int) ; k += blockDim.x)
        idest[k] = isource[k];
        
    TIMER_TOC(tid_memcpy)
        
}


    
__device__ inline void cuda_memcpy_old(void *dest, void *source, int count) {

    int k;
    volatile int *idest = (int *)dest, *isource = (int *)source;
    
    TIMER_TIC
    
    /* Copy the data in chunks of sizeof(int). */
    for(k = 0 + threadIdx.x ; k < count/sizeof(int) ; k += cuda_frame)
        idest[k] = isource[k];
        
    TIMER_TOC(tid_memcpy)
        
}


/**
 * @brief Sum two vectors in a strided way.
 *
 * @param a Pointer to destination memory.
 * @param b Pointer to source memory.
 * @param count Number of floats to sum.
 *
 * Computes @c a[k] += b[k] for k=1..count.
 */
 
__device__ inline void cuda_sum(float *a, float *b, int count) {

    int i, j, k, icount = count / cuda_frame / cuda_sum_chunk;
    float chunk[cuda_memcpy_chunk];
    int threadID = threadIdx.x;
    
    TIMER_TIC
    
    /* Copy the data in chunks of sizeof(int). */
    for(k = 0 ; k < icount ; k += 1) {
        #pragma unroll
        for(j = 0 ; j < cuda_sum_chunk ; j++) {
            i = (cuda_sum_chunk*k+j)*cuda_frame + threadID;
            chunk[j] = a[i] + b[i];
        }
        #pragma unroll
        for(j = 0 ; j < cuda_sum_chunk ; j++)
            a[ (cuda_sum_chunk*k+j)*cuda_frame + threadID ] = chunk[j];
    }
    for(k = cuda_sum_chunk*cuda_frame*icount + threadID ; k < count ; k += cuda_frame)
        a[k] += b[k];
        
    TIMER_TOC(tid_update)
        
}
    
    
    
    /**
 * @brief Sort the given data w.r.t. the lowest 16 bits in decending order.
 *
 * @param a The array to sort.
 * @param count The number of elements.
 */
 
__device__ inline void cuda_sort_descending(unsigned int *a, int count) {

    
    int i, j, k, threadID = threadIdx.x;
    int hi, lo, ind, jnd;
    unsigned int swap_i, swap_j;

    TIMER_TIC

    /* Sort using normalized bitonic sort. */
    for(k = 1 ; k < count ; k *= 2) {
    
        /* First step. */
        for(i = threadID ;  i < count ; i += blockDim.x) {
            hi = i & ~(k-1); lo = i & (k-1);
            ind = i + hi; jnd = 2*(hi+k) - lo - 1;
            swap_i =(jnd < count) ? a[ind] : 0;
            swap_j =(jnd < count) ? a[jnd] : 0;
            if ((swap_i & 0xffff) <(swap_j & 0xffff)) {
                a[ind] = swap_j;
                a[jnd] = swap_i;
            }
        }
            
        /* Let that last step sink in. */
            __syncthreads();
    
        /* Second step(s). */
        for(j = k/2 ; j > 0 ; j /= 2) {
            for(i = threadID ;  i < count ; i += blockDim.x) {
                hi = i & ~(j-1);
                ind = i + hi; jnd = ind + j;
                swap_i =(jnd < count) ? a[ind] : 0;
                swap_j =(jnd < count) ? a[jnd] : 0;
                if ((swap_i & 0xffff) <(swap_j & 0xffff)) {
                    a[ind] = swap_j;
                    a[jnd] = swap_i;
                }
            }
                __syncthreads();
        }
            
    }
        
    TIMER_TOC(tid_sort)

        
}

    


    
/**
 * @brief Sort the given data w.r.t. the lowest 16 bits in ascending order.
 *
 * @param a The array to sort.
 * @param count The number of elements.
 */
 
__device__ void cuda_sort_ascending(unsigned int *a, int count) {

    int i, j, k, threadID = threadIdx.x;
    int hi[2], lo[2], ind[2], jnd[2];
    unsigned int swap_i[2], swap_j[2];

    TIMER_TIC

    /* Sort using normalized bitonic sort. */
    for(k = 1 ; k < count ; k *= 2) {
    
        /* First step. */
        for(i = threadID ;  i < count ; i += 2*cuda_frame) {
            hi[0] = i & ~(k-1); lo[0] = i & (k-1);
            hi[1] = (i + cuda_frame) & ~(k-1); lo[1] = (i + cuda_frame) & (k-1);
            ind[0] = i + hi[0]; jnd[0] = 2*(hi[0]+k) - lo[0] - 1;
            ind[1] = i + cuda_frame + hi[1]; jnd[1] = 2*(hi[1]+k) - lo[1] - 1;
            swap_i[0] =(jnd[0] < count) ? a[ind[0]] : 0;
            swap_i[1] =(jnd[1] < count) ? a[ind[1]] : 0;
            swap_j[0] =(jnd[0] < count) ? a[jnd[0]] : 0;
            swap_j[1] =(jnd[1] < count) ? a[jnd[1]] : 0;
            if ((swap_i[0] & 0xffff) >(swap_j[0] & 0xffff)) {
                a[ind[0]] = swap_j[0];
                a[jnd[0]] = swap_i[0];
            }
            if ((swap_i[1] & 0xffff) >(swap_j[1] & 0xffff)) {
                a[ind[1]] = swap_j[1];
                a[jnd[1]] = swap_i[1];
            }
        }
            
        /* Let that last step sink in. */
        // __threadfence_block();
    
        /* Second step(s). */
        for(j = k/2 ; j > 0 ; j /= 2) {
            for(i = threadID ;  i < count ; i += 2*cuda_frame) {
                hi[0] = i & ~(j-1);
                hi[1] = (i + cuda_frame) & ~(j-1);
                ind[0] = i + hi[0]; jnd[0] = ind[0] + j;
                ind[1] = i + cuda_frame + hi[1]; jnd[1] = ind[1] + j;
                swap_i[0] =(jnd[0] < count) ? a[ind[0]] : 0;
                swap_i[1] =(jnd[1] < count) ? a[ind[1]] : 0;
                swap_j[0] =(jnd[0] < count) ? a[jnd[0]] : 0;
                swap_j[1] =(jnd[1] < count) ? a[jnd[1]] : 0;
                if ((swap_i[0] & 0xffff) >(swap_j[0] & 0xffff)) {
                    a[ind[0]] = swap_j[0];
                    a[jnd[0]] = swap_i[0];
                }
                if ((swap_i[1] & 0xffff) >(swap_j[1] & 0xffff)) {
                    a[ind[1]] = swap_j[1];
                    a[jnd[1]] = swap_i[1];
                }
            }
            // __threadfence_block();
        }
            
    }
        
    TIMER_TOC(tid_sort)
        
}

int engine_cuda_texture_init(hipTextureObject_t *tex, hipArray_t &arr) {
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arr;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.normalizedCoords = false;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.readMode = hipReadModeElementType;

    cuda_safe_call(hipCreateTextureObject(tex, &resDesc, &texDesc, NULL));

    return engine_err_ok;
}

int engine_cuda_texture_finalize(hipTextureObject_t tex) {
    cuda_safe_call(hipDestroyTextureObject(tex));
    
    return engine_err_ok;
}

int engine_cuda_build_pots_pack(Potential **pots, 
                                int nr_pots, 
                                std::vector<int> &pind, 
                                std::vector<float> &pot_alpha, 
                                std::vector<float> &pot_c,
                                std::vector<float> &pot_dataf, 
                                std::vector<int> &pot_datai, 
                                std::vector<float> &dpd_cf, 
                                std::vector<float> &dpd_dataf, 
                                std::vector<int> &dpd_datai, 
                                int &max_coeffs, 
                                int &max_pots, 
                                int &max_dpds, 
                                int &num_pots) 
{
    int i, j;
    pind = std::vector<int>(ENGINE_CUDA_PIND_WIDTH * nr_pots, 0);
    std::vector<Potential*> pots_unique(1, 0);

    /* Init the null potential. */
    if((pots_unique[0] = (struct Potential *)alloca(sizeof(struct Potential))) == NULL)
        return error(engine_err_malloc);
    pots_unique[0]->alpha[0] = pots_unique[0]->alpha[1] = pots_unique[0]->alpha[2] = pots_unique[0]->alpha[3] = 0.0f;
    pots_unique[0]->a = 0.0; pots_unique[0]->b = FLT_MAX;
    pots_unique[0]->flags = POTENTIAL_NONE;
    pots_unique[0]->n = 0;
    if((pots_unique[0]->c = (FPTYPE *)alloca(sizeof(float) * potential_chunk)) == NULL)
        return error(engine_err_malloc);
    bzero(pots_unique[0]->c, sizeof(float) * potential_chunk);
    
    /* Start by identifying the unique potentials in the engine. */
    max_pots = 1;
    max_dpds = 0;
    num_pots = 0;
    for(i = 0 ; i < nr_pots ; i++) {
    
        /* Skip if there is no potential or no parts of this type. */
        if(pots[i] == NULL)
            continue;
        
        num_pots++;

        /* Check this potential against previous potentials. */
        for(j = 0 ; j < pots_unique.size() && pots[i] != pots_unique[j] ; j++);
        if(j < pots_unique.size())
            continue;

        /* Store this potential and the number of coefficient entries it has. */
        pots_unique.push_back(pots[i]);

        std::vector<Potential*> pots_const;
        if(pots[i]->kind == POTENTIAL_KIND_COMBINATION && pots[i]->flags & POTENTIAL_SUM) {
            pots_const = pots[i]->constituents();
        } 
        else {
            pots_const = {pots[i]};
        }
        int nr_pots_i = 0;
        int nr_dpds_i = 0;
        for(auto &p : pots_const) {
            if(p->kind == POTENTIAL_KIND_DPD) 
                nr_dpds_i++;
            else if(p->kind == POTENTIAL_KIND_POTENTIAL) 
                nr_pots_i++;
        }
        max_dpds = std::max(max_dpds, nr_dpds_i);
        max_pots = std::max(max_pots, nr_pots_i);
    
    }

    std::vector<Potential*> pots_flat_unique(pots_unique.size() * max_pots, 0);
    std::vector<DPDPotential*> dpds_flat_unique(pots_unique.size() * max_dpds, 0);
    max_coeffs = 1;

    for(i = 0; i < pots_unique.size(); i++) {
        std::vector<Potential*> pots_const;
        if(pots_unique[i]->kind == POTENTIAL_KIND_COMBINATION && pots_unique[i]->flags & POTENTIAL_SUM) {
            pots_const = pots_unique[i]->constituents();
        } 
        else {
            pots_const = {pots_unique[i]};
        }
        for(j = 0; j < pots_const.size(); j++) {
            if(pots_const[j]->kind == POTENTIAL_KIND_DPD) 
                dpds_flat_unique[max_dpds * i + j] = (DPDPotential*)pots_const[j];
            else if(pots_const[j]->kind == POTENTIAL_KIND_POTENTIAL) {
                pots_flat_unique[max_pots * i + j] = pots_const[j];
                max_coeffs = std::max(max_coeffs, pots_const[j]->n + 1);
            }
        }
    }

    /* Pack the potential matrices. */
    for(i = 0 ; i < nr_pots ; i++) {
        if(pots[i] != NULL) {
            for(j = 0 ; j < pots_unique.size() && pots_unique[j] != pots[i] ; j++);

            std::vector<Potential*> pots_const;
            if(pots_unique[j]->kind == POTENTIAL_KIND_COMBINATION && pots_unique[j]->flags & POTENTIAL_SUM) {
                pots_const = pots_unique[j]->constituents();
            } 
            else {
                pots_const = {pots_unique[j]};
            }
            int nr_dpds_i = 0;
            int nr_pots_i = 0;
            for(auto &p : pots_const) 
                if(p->kind == POTENTIAL_KIND_DPD) 
                    nr_dpds_i++;
                else if(p->kind == POTENTIAL_KIND_POTENTIAL) 
                    nr_pots_i++;

            pind[i * ENGINE_CUDA_PIND_WIDTH    ] = j;
            pind[i * ENGINE_CUDA_PIND_WIDTH + 1] = nr_pots_i;
            pind[i * ENGINE_CUDA_PIND_WIDTH + 2] = nr_dpds_i;
        }
    }

    pot_alpha = std::vector<float>(pots_flat_unique.size() * ENGINE_CUDA_POT_WIDTH_ALPHA, 0);
    pot_c     = std::vector<float>(pots_flat_unique.size() * potential_chunk * max_coeffs, 0);
    pot_dataf = std::vector<float>(pots_flat_unique.size() * ENGINE_CUDA_POT_WIDTH_DATAF, 0);
    pot_datai = std::vector<int>(pots_flat_unique.size() * ENGINE_CUDA_POT_WIDTH_DATAI, 0);
    dpd_cf    = std::vector<float>(dpds_flat_unique.size() * ENGINE_CUDA_DPD_WIDTH_CF, 0);
    dpd_dataf = std::vector<float>(dpds_flat_unique.size() * ENGINE_CUDA_DPD_WIDTH_DATAF, 0);
    dpd_datai = std::vector<int>(dpds_flat_unique.size() * ENGINE_CUDA_DPD_WIDTH_DATAI, 0);

    // Pack the potentials
    for(i = 0; i < pots_flat_unique.size(); i++) { 
        Potential *p = pots_flat_unique[i];
        if(!p) 
            continue;
        pot_alpha[i * ENGINE_CUDA_POT_WIDTH_ALPHA    ] = p->alpha[0];
        pot_alpha[i * ENGINE_CUDA_POT_WIDTH_ALPHA + 1] = p->alpha[1];
        pot_alpha[i * ENGINE_CUDA_POT_WIDTH_ALPHA + 2] = p->alpha[2];
        for(j = 0; j < potential_chunk * (p->n + 1); j++) 
            pot_c[i * potential_chunk * max_coeffs + j] = p->c[j];
        pot_dataf[i * ENGINE_CUDA_POT_WIDTH_DATAF    ] = p->a;
        pot_dataf[i * ENGINE_CUDA_POT_WIDTH_DATAF + 1] = p->b;
        pot_dataf[i * ENGINE_CUDA_POT_WIDTH_DATAF + 2] = p->r0_plusone;
        pot_dataf[i * ENGINE_CUDA_POT_WIDTH_DATAF + 3] = p->offset[0];
        pot_dataf[i * ENGINE_CUDA_POT_WIDTH_DATAF + 4] = p->offset[1];
        pot_dataf[i * ENGINE_CUDA_POT_WIDTH_DATAF + 5] = p->offset[2];
        pot_datai[i * ENGINE_CUDA_POT_WIDTH_DATAI    ] = p->flags;
        pot_datai[i * ENGINE_CUDA_POT_WIDTH_DATAI + 1] = p->n;
    }
    for(i = 0; i < dpds_flat_unique.size(); i++) {
        DPDPotential *p = dpds_flat_unique[i];
        if(!p) 
            continue;
        dpd_cf[i * ENGINE_CUDA_DPD_WIDTH_CF    ] = p->alpha;
        dpd_cf[i * ENGINE_CUDA_DPD_WIDTH_CF + 1] = p->gamma;
        dpd_cf[i * ENGINE_CUDA_DPD_WIDTH_CF + 2] = p->sigma;
        dpd_dataf[i * ENGINE_CUDA_DPD_WIDTH_DATAF    ] = p->a;
        dpd_dataf[i * ENGINE_CUDA_DPD_WIDTH_DATAF + 1] = p->b;
        dpd_datai[i * ENGINE_CUDA_DPD_WIDTH_DATAI] = p->flags;
    }

    return engine_err_ok;
}

extern "C" int engine_cuda_boundary_conditions_load(struct engine *e) {

    BoundaryCondition bcs[] = {
        e->boundary_conditions.left, 
        e->boundary_conditions.right, 
        e->boundary_conditions.front, 
        e->boundary_conditions.back, 
        e->boundary_conditions.bottom, 
        e->boundary_conditions.top
    };

    hipChannelFormatDesc channelDesc_int = hipCreateChannelDesc<int>();
    hipChannelFormatDesc channelDesc_float = hipCreateChannelDesc<float>();

    int pots_bcs_max[6];
    int dpds_bcs_max[6];

    for(int bi = 0; bi < 6; bi++) {

        // GENERATE

        std::vector<int> pind;
        std::vector<float> pot_alpha;
        std::vector<float> pot_c;
        std::vector<float> pot_dataf;
        std::vector<int> pot_datai;
        std::vector<float> dpd_cf;
        std::vector<float> dpd_dataf;
        std::vector<int> dpd_datai;
        int max_coeffs;
        int max_pots;
        int max_dpds;
        int num_pots;

        cuda_safe_call_e(engine_cuda_build_pots_pack(
            bcs[bi].potenntials, e->max_type, 
            pind, pot_alpha, pot_c, pot_dataf, pot_datai, 
            dpd_cf, dpd_dataf, dpd_datai, 
            max_coeffs, max_pots, max_dpds, num_pots), engine_err_ok) ;

        pots_bcs_max[bi] = max_pots;
        dpds_bcs_max[bi] = max_dpds;

        for(int did = 0 ; did < e->nr_devices ; did++) {
            cuda_safe_call(hipSetDevice(e->devices[did]));

            cuda_safe_call(hipMalloc(&pind_bcs_cuda[did][bi], sizeof(int) * pind.size()));
            cuda_safe_call(hipMemcpy(pind_bcs_cuda[did][bi], pind.data(), sizeof(int) * pind.size(), hipMemcpyHostToDevice));

            cuda_safe_call(hipMallocArray(&cuda_pot_bcs_alpha[did][bi], &channelDesc_float, ENGINE_CUDA_POT_WIDTH_ALPHA, pot_alpha.size() / ENGINE_CUDA_POT_WIDTH_ALPHA, 0));
            cuda_safe_call(hipMemcpyToArray(cuda_pot_bcs_alpha[did][bi], 0, 0, pot_alpha.data(), sizeof(float) * pot_alpha.size(), hipMemcpyHostToDevice));
            cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_bcs_alpha[did][bi], cuda_pot_bcs_alpha[did][bi]), engine_err_ok);

            cuda_safe_call(hipMallocArray(&cuda_pot_bcs_c[did][bi], &channelDesc_float, potential_chunk * max_coeffs, pot_c.size() / (potential_chunk * max_coeffs), 0));
            cuda_safe_call(hipMemcpyToArray(cuda_pot_bcs_c[did][bi], 0, 0, pot_c.data(), sizeof(float) * pot_c.size(), hipMemcpyHostToDevice));
            cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_bcs_c[did][bi], cuda_pot_bcs_c[did][bi]), engine_err_ok);

            cuda_safe_call(hipMallocArray(&cuda_pot_bcs_dataf[did][bi], &channelDesc_float, ENGINE_CUDA_POT_WIDTH_DATAF, pot_dataf.size() / ENGINE_CUDA_POT_WIDTH_DATAF, 0));
            cuda_safe_call(hipMemcpyToArray(cuda_pot_bcs_dataf[did][bi], 0, 0, pot_dataf.data(), sizeof(float) * pot_dataf.size(), hipMemcpyHostToDevice));
            cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_bcs_dataf[did][bi], cuda_pot_bcs_dataf[did][bi]), engine_err_ok);

            cuda_safe_call(hipMallocArray(&cuda_pot_bcs_datai[did][bi], &channelDesc_int, ENGINE_CUDA_POT_WIDTH_DATAI, pot_datai.size() / ENGINE_CUDA_POT_WIDTH_DATAI, 0));
            cuda_safe_call(hipMemcpyToArray(cuda_pot_bcs_datai[did][bi], 0, 0, pot_datai.data(), sizeof(int) * pot_datai.size(), hipMemcpyHostToDevice));
            cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_bcs_datai[did][bi], cuda_pot_bcs_datai[did][bi]), engine_err_ok);

            cuda_safe_call(hipMallocArray(&cuda_dpd_bcs_cfs[did][bi], &channelDesc_float, ENGINE_CUDA_DPD_WIDTH_CF, dpd_cf.size() / ENGINE_CUDA_DPD_WIDTH_CF, 0));
            cuda_safe_call(hipMemcpyToArray(cuda_dpd_bcs_cfs[did][bi], 0, 0, dpd_cf.data(), sizeof(float) * dpd_cf.size(), hipMemcpyHostToDevice));
            cuda_safe_call_e(engine_cuda_texture_init(&tex_dpd_bcs_cfs[did][bi], cuda_dpd_bcs_cfs[did][bi]), engine_err_ok);

            cuda_safe_call(hipMallocArray(&cuda_dpd_bcs_dataf[did][bi], &channelDesc_float, ENGINE_CUDA_DPD_WIDTH_DATAF, dpd_dataf.size() / ENGINE_CUDA_DPD_WIDTH_DATAF, 0));
            cuda_safe_call(hipMemcpyToArray(cuda_dpd_bcs_dataf[did][bi], 0, 0, dpd_dataf.data(), sizeof(float) * dpd_dataf.size(), hipMemcpyHostToDevice));
            cuda_safe_call_e(engine_cuda_texture_init(&tex_dpd_bcs_dataf[did][bi], cuda_dpd_bcs_dataf[did][bi]), engine_err_ok);

            cuda_safe_call(hipMallocArray(&cuda_dpd_bcs_datai[did][bi], &channelDesc_int, ENGINE_CUDA_DPD_WIDTH_DATAI, dpd_datai.size() / ENGINE_CUDA_DPD_WIDTH_DATAI, 0));
            cuda_safe_call(hipMemcpyToArray(cuda_dpd_bcs_datai[did][bi], 0, 0, dpd_datai.data(), sizeof(int) * dpd_datai.size(), hipMemcpyHostToDevice));
            cuda_safe_call_e(engine_cuda_texture_init(&tex_dpd_bcs_datai[did][bi], cuda_dpd_bcs_datai[did][bi]), engine_err_ok);

        }
    }

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call(hipSetDevice(e->devices[did]));

        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_pind_bcs), pind_bcs_cuda[did], 6 * sizeof(int *), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_bcs_alpha), &tex_pot_bcs_alpha[did], 6 * sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_bcs_c), &tex_pot_bcs_c[did], 6 * sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_bcs_dataf), &tex_pot_bcs_dataf[did], 6 * sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_bcs_datai), &tex_pot_bcs_datai[did], 6 * sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_dpd_bcs_cfs), &tex_dpd_bcs_cfs[did], 6 * sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_dpd_bcs_dataf), &tex_dpd_bcs_dataf[did], 6 * sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_dpd_bcs_datai), &tex_dpd_bcs_datai[did], 6 * sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_pots_bcs_max), pots_bcs_max, 6 * sizeof(int), 0, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_dpds_bcs_max), dpds_bcs_max, 6 * sizeof(int), 0, hipMemcpyHostToDevice));

        cuda_bcs_dev = cuda::BoundaryConditions(e->boundary_conditions);

        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_bcs), &cuda_bcs_dev, sizeof(cuda::BoundaryConditions), 0, hipMemcpyHostToDevice));

    }

    return engine_err_ok;
}

/**
 * @brief Finalize boundary conditions on device. 
 * 
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
int engine_cuda_boundary_conditions_finalize(struct engine *e) {

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call(hipSetDevice(e->devices[did]));

        for(int bi = 0; bi < 6; bi++) {

            cuda_safe_call(hipFree(pind_bcs_cuda[did][bi]));

            cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_bcs_alpha[did][bi]), engine_err_ok);
            cuda_safe_call(hipFreeArray(cuda_pot_bcs_alpha[did][bi]));

            cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_bcs_c[did][bi]), engine_err_ok);
            cuda_safe_call(hipFreeArray(cuda_pot_bcs_c[did][bi]));

            cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_bcs_dataf[did][bi]), engine_err_ok);
            cuda_safe_call(hipFreeArray(cuda_pot_bcs_dataf[did][bi]));

            cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_bcs_datai[did][bi]), engine_err_ok);
            cuda_safe_call(hipFreeArray(cuda_pot_bcs_datai[did][bi]));

            cuda_safe_call_e(engine_cuda_texture_finalize(tex_dpd_bcs_cfs[did][bi]), engine_err_ok);
            cuda_safe_call(hipFreeArray(cuda_dpd_bcs_cfs[did][bi]));

            cuda_safe_call_e(engine_cuda_texture_finalize(tex_dpd_bcs_dataf[did][bi]), engine_err_ok);
            cuda_safe_call(hipFreeArray(cuda_dpd_bcs_dataf[did][bi]));

            cuda_safe_call_e(engine_cuda_texture_finalize(tex_dpd_bcs_datai[did][bi]), engine_err_ok);
            cuda_safe_call(hipFreeArray(cuda_dpd_bcs_datai[did][bi]));

        }

    }

    return engine_err_ok;
}

/**
 * @brief Refresh boundary conditions on device. Can be safely called while on device. 
 * 
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
extern "C" int cuda::engine_cuda_boundary_conditions_refresh(struct engine *e) {
    
    if(engine_cuda_boundary_conditions_finalize(e) < 0)
        return error(engine_err);

    if(engine_cuda_boundary_conditions_load(e) < 0)
        return error(engine_err);

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call(hipSetDevice(e->devices[did]));

        cuda_safe_call(hipDeviceSynchronize());

    }

    return engine_err_ok;
}


__device__ void cuda::potential_eval_r_cuda(struct TissueForge::Potential *p, FPTYPE r, FPTYPE *e, FPTYPE *f) {

    int ind, k;
    FPTYPE x, ee, eff, *c;
    
    TIMER_TIC

    /* compute the index */
    ind = fmaxf(0.0f, p->alpha[0] + r * (p->alpha[1] + r * p->alpha[2]));

    /* get the table offset */
    c = &(p->c[ind * potential_chunk]);

    /* adjust x to the interval */
    x = (r - c[0]) * c[1];

    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    for(k = 4 ; k < potential_chunk ; k++) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
        }

    /* store the result */
    *e = ee; *f = eff * c[1];

    TIMER_TOC(tid_potential)

}

__device__ void cuda::potential_eval_cuda(struct TissueForge::Potential *p, float r2, float *e, float *f) {

    int ind, k;
    float x, ee, eff, *c, ir, r;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    /* compute the interval index */
    ind = fmaxf(0.0f, p->alpha[0] + r * (p->alpha[1] + r * p->alpha[2]));
    
    /* get the table offset */
    c = &(p->c[ind * potential_chunk]);
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    #pragma unroll
    for(k = 4 ; k < potential_chunk ; k++) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
    }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;
        
    TIMER_TOC(tid_potential)
        
}

__device__ void potential_eval_cuda(cuda::PotentialData p, float r2, float *e, float *f) {

    int ind, k;
    float x, ee, eff, *c, ir, r;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    /* compute the interval index */
    ind = fmaxf(0.0f, p.alpha.x + r * (p.alpha.y + r * p.alpha.z));
    
    /* get the table offset */
    c = &(p.c[ind * potential_chunk]);
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    #pragma unroll
    for(k = 4 ; k < potential_chunk ; k++) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
    }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;
        
    TIMER_TOC(tid_potential)
        
}

__device__ void potential_eval_cuda(int pind, bool iscluster, float r2, float *e, float *f) {

    int ind, k;
    float x, ee, eff, ir, r;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    /* compute the interval index */
    float3 alpha;
    if(iscluster) {
        alpha.x = tex2D<float>(cuda_tex_pot_cluster_alpha, 0, pind);
        alpha.y = tex2D<float>(cuda_tex_pot_cluster_alpha, 1, pind);
        alpha.z = tex2D<float>(cuda_tex_pot_cluster_alpha, 2, pind);
    } 
    else {
        alpha.x = tex2D<float>(cuda_tex_pot_alpha, 0, pind);
        alpha.y = tex2D<float>(cuda_tex_pot_alpha, 1, pind);
        alpha.z = tex2D<float>(cuda_tex_pot_alpha, 2, pind);
    }
    ind = fmaxf(0.0f, alpha.x + r * (alpha.y + r * alpha.z));
    
    /* get the table offset */
    float c[potential_chunk];
    if(iscluster) {
        for(k = 0; k < potential_chunk; k++) 
            c[k] = tex2D<float>(cuda_tex_pot_cluster_c, ind * potential_chunk + k, pind);
    } 
    else {
        for(k = 0; k < potential_chunk; k++) 
            c[k] = tex2D<float>(cuda_tex_pot_c, ind * potential_chunk + k, pind);
    }
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    #pragma unroll
    for(k = 4 ; k < potential_chunk ; k++) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
    }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;
        
    TIMER_TOC(tid_potential)
        
}

/** 
 * @brief Evaluates the given potential at the given point (interpolated).
 *
 * @param p The #potential to be evaluated.
 * @param ri Radius of the ith particle. 
 * @param rj Radius of the jth particle. 
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */
__device__ inline void potential_eval_ex_cuda (cuda::PotentialData p, float ri, float rj, float r2, float *e, float *f) {

    int ind, k;
    float x, ee, eff, *c, ir, r;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    if(p.flags & POTENTIAL_SCALED) {
        r = r / (ri + rj);
    }
    else if(p.flags & POTENTIAL_SHIFTED) {
        r = r - (ri + rj) + p.w.z;
    }
    
    // cutoff min value, eval at lowest func interpolation.
    r = r < p.w.x ? p.w.x : r;

    if(r > p.w.y) {
        *e = 0.f;
        *f = 0.f;
        return;
    }
    
    /* compute the interval index */
    ind = fmaxf(0.0f, p.alpha.x + r * (p.alpha.y + r * p.alpha.z));

    if(ind > p.n) {
        *e = 0.f;
        *f = 0.f;
        return;
    }
    
    /* get the table offset */
    c = &(p.c[ind * potential_chunk]);
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    #pragma unroll
    for(k = 4 ; k < potential_chunk ; k++) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
    }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;

    TIMER_TOC(tid_potential)
        
}

/** 
 * @brief Evaluates the given potential at the given point (interpolated).
 *
 * @param p The #potential to be evaluated.
 * @param ri Radius of the ith particle. 
 * @param rj Radius of the jth particle. 
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */
__device__ inline void potential_eval_ex_cuda (int pind, bool iscluster, float ri, float rj, float r2, float *e, float *f) {

    int ind, k;
    float x, ee, eff, ir, r;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    int p_flags, n;
    float a, b, r0_plusone;
    if(iscluster) {
        p_flags    = tex2D<int>(cuda_tex_pot_cluster_datai, 0, pind);
        n          = tex2D<int>(cuda_tex_pot_cluster_datai, 1, pind);
        a          = tex2D<float>(cuda_tex_pot_cluster_dataf, 0, pind);
        b          = tex2D<float>(cuda_tex_pot_cluster_dataf, 1, pind);
        r0_plusone = tex2D<float>(cuda_tex_pot_cluster_dataf, 2, pind);
    } 
    else {
        p_flags    = tex2D<int>(cuda_tex_pot_datai, 0, pind);
        n          = tex2D<int>(cuda_tex_pot_datai, 1, pind);
        a          = tex2D<float>(cuda_tex_pot_dataf, 0, pind);
        b          = tex2D<float>(cuda_tex_pot_dataf, 1, pind);
        r0_plusone = tex2D<float>(cuda_tex_pot_dataf, 2, pind);
    }
    
    if(p_flags & POTENTIAL_SCALED) {
        r = r / (ri + rj);
    }
    else if(p_flags & POTENTIAL_SHIFTED) {
        r = r - (ri + rj) + r0_plusone;
    }
    
    // cutoff min value, eval at lowest func interpolation.
    r = r < a ? a : r;

    if(r > b) {
        *e = 0.f;
        *f = 0.f;
        return;
    }
    
    /* compute the interval index */
    float3 alpha;
    if(iscluster) {
        alpha.x = tex2D<float>(cuda_tex_pot_cluster_alpha, 0, pind);
        alpha.y = tex2D<float>(cuda_tex_pot_cluster_alpha, 1, pind);
        alpha.z = tex2D<float>(cuda_tex_pot_cluster_alpha, 2, pind);
    } 
    else {
        alpha.x = tex2D<float>(cuda_tex_pot_alpha, 0, pind);
        alpha.y = tex2D<float>(cuda_tex_pot_alpha, 1, pind);
        alpha.z = tex2D<float>(cuda_tex_pot_alpha, 2, pind);
    }
    
    ind = fmaxf(0.0f, alpha.x + r * (alpha.y + r * alpha.z));

    if(ind > n) {
        *e = 0.f;
        *f = 0.f;
        return;
    }
    
    /* get the table offset */
    float c[potential_chunk];
    if(iscluster) {
        for(k = 0; k < potential_chunk; k++) 
            c[k] = tex2D<float>(cuda_tex_pot_cluster_c, ind * potential_chunk + k, pind);
    } 
    else {
        for(k = 0; k < potential_chunk; k++) 
            c[k] = tex2D<float>(cuda_tex_pot_c, ind * potential_chunk + k, pind);
    }
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    #pragma unroll
    for(k = 4 ; k < potential_chunk ; k++) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
    }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;

    TIMER_TOC(tid_potential)
        
}

/** 
 * @brief Evaluates the given potential at the given point (interpolated).
 *
 * @param p The #potential to be evaluated.
 * @param ri Radius of the ith particle. 
 * @param rj Radius of the jth particle. 
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */
__device__ inline void potential_eval_ex_cuda (int pind, int bid, float ri, float rj, float r2, float *e, float *f) {

    int ind, k;
    float x, ee, eff, ir, r;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    int p_flags      = tex2D<int>(cuda_tex_pot_bcs_datai[bid], 0, pind);
    int n            = tex2D<int>(cuda_tex_pot_bcs_datai[bid], 1, pind);
    float a          = tex2D<float>(cuda_tex_pot_bcs_dataf[bid], 0, pind);
    float b          = tex2D<float>(cuda_tex_pot_bcs_dataf[bid], 1, pind);
    float r0_plusone = tex2D<float>(cuda_tex_pot_bcs_dataf[bid], 2, pind);
    
    if(p_flags & POTENTIAL_SCALED) {
        r = r / (ri + rj);
    }
    else if(p_flags & POTENTIAL_SHIFTED) {
        r = r - (ri + rj) + r0_plusone;
    }
    
    // cutoff min value, eval at lowest func interpolation.
    r = r < a ? a : r;

    if(r > b) {
        *e = 0.f;
        *f = 0.f;
        return;
    }
    
    /* compute the interval index */
    float3 alpha;
    alpha.x = tex2D<float>(cuda_tex_pot_bcs_alpha[bid], 0, pind);
    alpha.y = tex2D<float>(cuda_tex_pot_bcs_alpha[bid], 1, pind);
    alpha.z = tex2D<float>(cuda_tex_pot_bcs_alpha[bid], 2, pind);
    
    ind = fmaxf(0.0f, alpha.x + r * (alpha.y + r * alpha.z));

    if(ind > n) {
        *e = 0.f;
        *f = 0.f;
        return;
    }
    
    /* get the table offset */
    float c[potential_chunk];
    for(k = 0; k < potential_chunk; k++) 
        c[k] = tex2D<float>(cuda_tex_pot_bcs_c[bid], ind * potential_chunk + k, pind);
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    #pragma unroll
    for(k = 4 ; k < potential_chunk ; k++) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
    }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;

    TIMER_TOC(tid_potential)
        
}


__device__ inline void dpd_eval_cuda(cuda::DPDPotentialData pot, cuda::Particle pi, cuda::Particle pj, float3 dx, float r2, float *e, float *fi, float *fj) {

    float delta = rsqrtf(cuda_dt);
    
    float r = sqrtf(r2);
    float ro = r < FLT_MIN ? FLT_MIN : r;
    
    r = pot.flags & POTENTIAL_SHIFTED ? r - (pi.v.w + pj.v.w) : r;

    if(r > pot.w.y) {
        *e = 0.f;
        return;
    }
    r = r >= pot.w.x ? r : pot.w.x;
    
    // unit vector
    float3 unit_vec{dx.x / ro, dx.y / ro, dx.z / ro};
    
    float3 v{pi.v.x - pj.v.x, pi.v.y - pj.v.y, pi.v.z - pj.v.z};
    
    // conservative force
    float omega_c = r < 0.f ?  1.f : (1 - r / pot.w.y);
    
    float fc = pot.dpd_cfs.x * omega_c;
    
    // dissapative force
    float omega_d = omega_c * omega_c;
    
    float fd = - pot.dpd_cfs.y * omega_d * (unit_vec.x * v.x + unit_vec.y * v.y + unit_vec.z * v.z);
    
    float fr = pot.dpd_cfs.z * omega_c * delta;
    
    float f = fc + fd + fr;
    
    fi[0] += f * unit_vec.x;
    fi[1] += f * unit_vec.y;
    fi[2] += f * unit_vec.z;
    fj[0] -= f * unit_vec.x;
    fj[1] -= f * unit_vec.y;
    fj[2] -= f * unit_vec.z;
    
    // TODO: correct energy
    *e = 0;

}

__device__ inline void dpd_eval_cuda(int pind, bool iscluster, cuda::Particle pi, cuda::Particle pj, float3 dx, float r2, float *e, float *fi, float *fj) {

    float delta = rsqrtf(cuda_dt);
    
    float r = sqrtf(r2);
    float ro = r < FLT_MIN ? FLT_MIN : r;

    int flags;
    float a, b;

    if(iscluster) {
        flags = tex2D<int>(cuda_tex_dpd_cluster_datai, 0, pind);
        a     = tex2D<float>(cuda_tex_dpd_cluster_dataf, 0, pind);
        b     = tex2D<float>(cuda_tex_dpd_cluster_dataf, 1, pind);
    } 
    else {
        flags = tex2D<int>(cuda_tex_dpd_datai, 0, pind);
        a     = tex2D<float>(cuda_tex_dpd_dataf, 0, pind);
        b     = tex2D<float>(cuda_tex_dpd_dataf, 1, pind);
    }
    
    r = flags & POTENTIAL_SHIFTED ? r - (pi.v.w + pj.v.w) : r;

    if(r > b) {
        *e = 0.f;
        return;
    }
    r = r >= a ? r : a;

    float alpha, gamma, sigma;

    if(iscluster) {
        alpha = tex2D<float>(cuda_tex_dpd_cluster_cfs, 0, pind);
        gamma = tex2D<float>(cuda_tex_dpd_cluster_cfs, 1, pind);
        sigma = tex2D<float>(cuda_tex_dpd_cluster_cfs, 2, pind);
    } 
    else {
        alpha = tex2D<float>(cuda_tex_dpd_cfs, 0, pind);
        gamma = tex2D<float>(cuda_tex_dpd_cfs, 1, pind);
        sigma = tex2D<float>(cuda_tex_dpd_cfs, 2, pind);
    }
    
    // unit vector
    float3 unit_vec{dx.x / ro, dx.y / ro, dx.z / ro};
    
    float3 v{pi.v.x - pj.v.x, pi.v.y - pj.v.y, pi.v.z - pj.v.z};
    
    // conservative force
    float omega_c = r < 0.f ?  1.f : (1 - r / b);
    
    float fc = alpha * omega_c;
    
    // dissapative force
    float omega_d = omega_c * omega_c;
    
    float fd = - gamma * omega_d * (unit_vec.x * v.x + unit_vec.y * v.y + unit_vec.z * v.z);
    
    float fr = sigma * omega_c * delta;
    
    float f = fc + fd + fr;
    
    fi[0] += f * unit_vec.x;
    fi[1] += f * unit_vec.y;
    fi[2] += f * unit_vec.z;
    fj[0] -= f * unit_vec.x;
    fj[1] -= f * unit_vec.y;
    fj[2] -= f * unit_vec.z;
    
    // TODO: correct energy
    *e = 0;

}


__device__ inline void dpd_boundary_eval_cuda(cuda::DPDPotentialData pot, cuda::Particle pi, float rj, float3 velocity, float3 dx, float r2, float *e, float *force) {

    float delta = rsqrtf(cuda_dt);
    
    float r = sqrtf(r2);
    float ro = r < FLT_MIN ? FLT_MIN : r;
    
    r = pot.flags & POTENTIAL_SHIFTED ? r - (pi.v.w + rj) : r;

    if(r > pot.w.y) {
        *e = 0.f;
        return;
    }
    r = r >= pot.w.x ? r : pot.w.x;
    
    // unit vector
    float3 unit_vec{dx.x / ro, dx.y / ro, dx.z / ro};
    
    float3 v{pi.v.x - velocity.x, pi.v.y - velocity.y, pi.v.z - velocity.z};
    
    // conservative force
    float omega_c = r < 0.f ?  1.f : (1 - r / pot.w.y);
    
    float fc = pot.dpd_cfs.x * omega_c;
    
    // dissapative force
    float omega_d = omega_c * omega_c;
    
    float fd = - pot.dpd_cfs.y * omega_d * (unit_vec.x * v.x + unit_vec.y * v.y + unit_vec.z * v.z);
    
    float fr = pot.dpd_cfs.z * omega_c * delta;
    
    float f = fc + fd + fr;
    
    force[0] += f * unit_vec.x;
    force[1] += f * unit_vec.y;
    force[2] += f * unit_vec.z;
    
    // TODO: correct energy
    *e = 0;

}

__device__ inline void dpd_boundary_eval_cuda(int pind, int bid, cuda::Particle pi, float rj, float3 velocity, float3 dx, float r2, float *e, float *force) {

    float delta = rsqrtf(cuda_dt);
    
    float r = sqrtf(r2);
    float ro = r < FLT_MIN ? FLT_MIN : r;
    
    int flags = tex2D<int>(cuda_tex_dpd_bcs_datai[bid], 0, pind);
    float a = tex2D<float>(cuda_tex_dpd_bcs_dataf[bid], 0, pind);
    float b = tex2D<float>(cuda_tex_dpd_bcs_dataf[bid], 1, pind);;
    
    r = flags & POTENTIAL_SHIFTED ? r - (pi.v.w + rj) : r;

    if(r > b) {
        *e = 0.f;
        return;
    }
    r = r >= a ? r : a;

    float3 dpd_cfs;
    dpd_cfs.x = tex2D<float>(cuda_tex_dpd_bcs_cfs[bid], 0, pind);
    dpd_cfs.y = tex2D<float>(cuda_tex_dpd_bcs_cfs[bid], 1, pind);
    dpd_cfs.z = tex2D<float>(cuda_tex_dpd_bcs_cfs[bid], 2, pind);
    
    // unit vector
    float3 unit_vec{dx.x / ro, dx.y / ro, dx.z / ro};
    
    float3 v{pi.v.x - velocity.x, pi.v.y - velocity.y, pi.v.z - velocity.z};
    
    // conservative force
    float omega_c = r < 0.f ?  1.f : (1 - r / b);
    
    float fc = dpd_cfs.x * omega_c;
    
    // dissapative force
    float omega_d = omega_c * omega_c;
    
    float fd = - dpd_cfs.y * omega_d * (unit_vec.x * v.x + unit_vec.y * v.y + unit_vec.z * v.z);
    
    float fr = dpd_cfs.z * omega_c * delta;
    
    float f = fc + fd + fr;
    
    force[0] += f * unit_vec.x;
    force[1] += f * unit_vec.y;
    force[2] += f * unit_vec.z;
    
    // TODO: correct energy
    *e = 0;

}


// Underlying evaluation call
__device__ inline void _potential_eval_super_ex_cuda_p(int pind, 
                                                    bool iscluster, 
                                                    cuda::Particle pi, 
                                                    cuda::Particle pj, 
                                                    float3 dx, 
                                                    float r2, 
                                                    float *epot, 
                                                    float *fi, 
                                                    float *fj) 
{
    int p_flags = tex2D<int>(iscluster ? cuda_tex_pot_cluster_datai : cuda_tex_pot_datai, 0, pind);
    if(p_flags & POTENTIAL_PERIODIC) {
        // Assuming elsewhere there's a corresponding potential in the opposite direction
        hipTextureObject_t tex = iscluster ? cuda_tex_pot_cluster_dataf : cuda_tex_pot_dataf;
        float3 offset;
        offset.x = tex2D<float>(tex, 3, pind);
        offset.y = tex2D<float>(tex, 4, pind);
        offset.z = tex2D<float>(tex, 5, pind);
        dx.x -= offset.x;
        dx.y -= offset.y;
        dx.z -= offset.z;
        r2 = dx.x * dx.x + dx.y * dx.y + dx.z * dx.z;
    }
    
    float e, f;

    /* update the forces if part in range */
    potential_eval_ex_cuda(pind, iscluster, pi.v.w, pj.v.w, r2, &e, &f);

    float w;
    w = f * dx.x;
    fi[0] -= w;
    fj[0] += w;
    w = f * dx.y;
    fi[1] -= w;
    fj[1] += w;
    w = f * dx.z;
    fi[2] -= w;
    fj[2] += w;
    
    /* tabulate the energy */
    *epot += e;
}


// Underlying evaluation call
__device__ inline void _potential_eval_super_ex_cuda_d(int pind, 
                                                    bool iscluster,
                                                    cuda::Particle pi, 
                                                    cuda::Particle pj, 
                                                    float3 dx, 
                                                    float r2, 
                                                    float *epot, 
                                                    float *fi, 
                                                    float *fj) 
{
    float e;

    /* update the forces if part in range */
    dpd_eval_cuda(pind, iscluster, pi, pj, dx, r2, &e, fi, fj);
    
    *epot += e;
}

template <bool iscluster> 
__device__ inline 
void potential_eval_super_ex_cuda(int pind, 
                                  int nr_pots, 
                                  int nr_dpds, 
                                  cuda::Particle pi, 
                                  cuda::Particle pj, 
                                  float3 dx, 
                                  float r2, 
                                  float *epot, 
                                  float *fi, 
                                  float *fj) 
{

    if(iscluster) {
        int stride = cuda_dpds_cluster_max;
        for(int i = 0; i < nr_dpds; i++) {
            _potential_eval_super_ex_cuda_d(stride * pind + i, true, pi, pj, dx, r2, epot, fi, fj);
        }

        stride = cuda_pots_cluster_max;
        for(int i = 0; i < nr_pots; i++) {
            _potential_eval_super_ex_cuda_p(stride * pind + i, true, pi, pj, dx, r2, epot, fi, fj);
        }
    } 
    else {
        int stride = cuda_dpds_max;
        for(int i = 0; i < nr_dpds; i++) {
            _potential_eval_super_ex_cuda_d(stride * pind + i, false, pi, pj, dx, r2, epot, fi, fj);
        }

        stride = cuda_pots_max;
        for(int i = 0; i < nr_pots; i++) {
            _potential_eval_super_ex_cuda_p(stride * pind + i, false, pi, pj, dx, r2, epot, fi, fj);
        }
    }

}

__device__ inline void _boundary_eval_cuda_ex_cuda_p(int pind, 
                                                     int bid, 
                                                     cuda::Particle part, 
                                                     cuda::BoundaryCondition bc, 
                                                     float3 dx, 
                                                     float r2, 
                                                     float *epot, 
                                                     float *force) 
{
    float e, f;

    /* update the forces if part in range */
    potential_eval_ex_cuda(pind, bid, part.v.w, bc.radius, r2, &e, &f);

    force[0] -= f * dx.x;
    force[1] -= f * dx.y;
    force[2] -= f * dx.z;
    
    /* tabulate the energy */
    *epot += e;
}


__device__ inline 
void _boundary_eval_cuda_ex_cuda_d(int pind, 
                                   int bid, 
                                   cuda::Particle part, 
                                   cuda::BoundaryCondition bc, 
                                   float3 dx, 
                                   float r2, 
                                   float *epot, 
                                   float *force) 
{
    float e;
    
    /* update the forces if part in range */
    dpd_boundary_eval_cuda(pind, bid, part, bc.radius, bc.velocity, dx, r2, &e, force);
    
    *epot += e;
}

__device__ inline 
void boundary_eval_cuda_ex_cuda(int pind, 
                                int nr_pots, 
                                int nr_dpds, 
                                int bid, 
                                cuda::Particle part, 
                                cuda::BoundaryCondition bc, 
                                float3 dx, 
                                float r, 
                                float *epot, 
                                float *force) 
{
    float r2 = r * r;

    for(int i = 0; i < nr_dpds; i++) {
        _boundary_eval_cuda_ex_cuda_d(cuda_dpds_bcs_max[bid] * pind + i, bid, part, bc, dx, r2, epot, force);
    }

    for(int i = 0; i < nr_pots; i++) {
        _boundary_eval_cuda_ex_cuda_p(cuda_pots_bcs_max[bid] * pind + i, bid, part, bc, dx, r2, epot, force);
    }
}

__device__ 
void boundary_eval_cuda(cuda::Particle part, float3 cell_dim, unsigned int cell_flags, float *force, float *epot) {
    
    float r;
    cuda::BoundaryCondition *bcs = cuda_bcs.bcs;
    
    if(cell_flags & cell_active_left) {
        int pind    = cuda_pind_bcs[0][ENGINE_CUDA_PIND_WIDTH * part.w.y    ];
        int nr_pots = cuda_pind_bcs[0][ENGINE_CUDA_PIND_WIDTH * part.w.y + 1];
        int nr_dpds = cuda_pind_bcs[0][ENGINE_CUDA_PIND_WIDTH * part.w.y + 2];
        if(nr_pots + nr_dpds > 0) {
            r = part.x.x;
            float3 dx{r, 0.f, 0.f};
            boundary_eval_cuda_ex_cuda(pind, nr_pots, nr_dpds, 0, part, bcs[0], dx, r, epot, force);
        }
    }
    
    if(cell_flags & cell_active_right) {
        int pind    = cuda_pind_bcs[1][ENGINE_CUDA_PIND_WIDTH * part.w.y    ];
        int nr_pots = cuda_pind_bcs[1][ENGINE_CUDA_PIND_WIDTH * part.w.y + 1];
        int nr_dpds = cuda_pind_bcs[1][ENGINE_CUDA_PIND_WIDTH * part.w.y + 2];
        if(nr_pots + nr_dpds > 0) {
            r = cell_dim.x - part.x.x;
            float3 dx{-r, 0.f, 0.f};
            boundary_eval_cuda_ex_cuda(pind, nr_pots, nr_dpds, 1, part, bcs[1], dx, r, epot, force);
        }
    }
    
    if(cell_flags & cell_active_front) {
        int pind    = cuda_pind_bcs[2][ENGINE_CUDA_PIND_WIDTH * part.w.y    ];
        int nr_pots = cuda_pind_bcs[2][ENGINE_CUDA_PIND_WIDTH * part.w.y + 1];
        int nr_dpds = cuda_pind_bcs[2][ENGINE_CUDA_PIND_WIDTH * part.w.y + 2];
        if(nr_pots + nr_dpds > 0) {
            r = part.x.y;
            float3 dx{0.f, r, 0.f};
            boundary_eval_cuda_ex_cuda(pind, nr_pots, nr_dpds, 2, part, bcs[2], dx, r, epot, force);
        }
    }
    
    if(cell_flags & cell_active_back) {
        int pind    = cuda_pind_bcs[3][ENGINE_CUDA_PIND_WIDTH * part.w.y    ];
        int nr_pots = cuda_pind_bcs[3][ENGINE_CUDA_PIND_WIDTH * part.w.y + 1];
        int nr_dpds = cuda_pind_bcs[3][ENGINE_CUDA_PIND_WIDTH * part.w.y + 2];
        if(nr_pots + nr_dpds > 0) {
            r = cell_dim.y - part.x.y;
            float3 dx{0.f, -r, 0.f};
            boundary_eval_cuda_ex_cuda(pind, nr_pots, nr_dpds, 3, part, bcs[3], dx, r, epot, force);
        }
    }
    
    if(cell_flags & cell_active_bottom) {
        int pind    = cuda_pind_bcs[4][ENGINE_CUDA_PIND_WIDTH * part.w.y    ];
        int nr_pots = cuda_pind_bcs[4][ENGINE_CUDA_PIND_WIDTH * part.w.y + 1];
        int nr_dpds = cuda_pind_bcs[4][ENGINE_CUDA_PIND_WIDTH * part.w.y + 2];
        if(nr_pots + nr_dpds > 0) {
            r = part.x.z;
            float3 dx{0.f, 0.f, r};
            boundary_eval_cuda_ex_cuda(pind, nr_pots, nr_dpds, 4, part, bcs[4], dx, r, epot, force);
        }
    }
    
    if(cell_flags & cell_active_top) {
        int pind    = cuda_pind_bcs[5][ENGINE_CUDA_PIND_WIDTH * part.w.y    ];
        int nr_pots = cuda_pind_bcs[5][ENGINE_CUDA_PIND_WIDTH * part.w.y + 1];
        int nr_dpds = cuda_pind_bcs[5][ENGINE_CUDA_PIND_WIDTH * part.w.y + 2];
        if(nr_pots + nr_dpds > 0) {
            r = cell_dim.z - part.x.z;
            float3 dx{0.f, 0.f, -r};
            boundary_eval_cuda_ex_cuda(pind, nr_pots, nr_dpds, 5, part, bcs[5], dx, r, epot, force);
        }
    }

}


__device__ inline 
void flux_fick_cuda(cuda::Flux flux, int i, float si, float sj, float *result) {
    *result *= flux.coef[i] * (si - sj);
}

__device__ inline 
void flux_secrete_cuda(cuda::Flux flux, int i, float si, float sj, float *result) {
    float q = flux.coef[i] * (si - flux.target[i]);
    float scale = q > 0.f;  // forward only, 1 if > 0, 0 if < 0.
    *result *= scale * q;
}

__device__ inline 
void flux_uptake_cuda(cuda::Flux flux, int i, float si, float sj, float *result) {
    float q = flux.coef[i] * (flux.target[i] - sj) * si;
    float scale = q > 0.f;
    *result *= scale * q;
}

__device__ inline 
void flux_eval_ex_cuda(cuda::Fluxes fluxes, float r, float *states_i, float *states_j, int type_i, int type_j, float *qvec_i) {

    // Do calculations

    float ssi, ssj;
    float q;
    
    cuda::Flux flux = fluxes.fluxes[0];
    float term = 1. - r / cuda_cutoff;
    term = term * term;

    int qind;
    
    for(int i = 0; i < flux.size; ++i) {

        if(type_i == flux.type_ids[i].a) {
            qind = flux.indices_a[i];
            ssi = states_i[qind];
            ssj = states_j[flux.indices_b[i]];
            q = - term;
        }
        else {
            qind = flux.indices_b[i];
            ssi = states_j[flux.indices_a[i]];
            ssj = states_i[qind];
            q = term;
        }

        switch(flux.kinds[i]) {
            case FLUX_FICK:
                flux_fick_cuda(flux, i, ssi, ssj, &q);
                break;
            case FLUX_SECRETE:
                flux_secrete_cuda(flux, i, ssi, ssj, &q);
                break;
            case FLUX_UPTAKE:
                flux_uptake_cuda(flux, i, ssi, ssj, &q);
                break;
            default:
                __builtin_unreachable();
        }

        qvec_i[qind] += q - 0.5 * flux.decay_coef[i] * states_i[qind];
    }
}

__device__ inline 
void flux_eval_ex_cuda(cuda::Fluxes fluxes, float r, float *states_i, float *states_j, int type_i, int type_j, float *qvec_i, float *qvec_j) {

    // Do calculations
    
    cuda::Flux flux = fluxes.fluxes[0];
    float term = 1. - r / cuda_cutoff;
    term = term * term;

    float *qi, *qj, *si, *sj;
    
    for(int i = 0; i < flux.size; ++i) {

        if(type_i == flux.type_ids[i].a) {
            si = states_i;
            sj = states_j;
            qi = qvec_i;
            qj = qvec_j;
        }
        else {
            si = states_j;
            sj = states_i;
            qi = qvec_j;
            qj = qvec_i;
        }
        
        float ssi = si[flux.indices_a[i]];
        float ssj = sj[flux.indices_b[i]];
        float q =  term;
        float mult;
        
        switch(flux.kinds[i]) {
            case FLUX_FICK:
                flux_fick_cuda(flux, i, ssi, ssj, &mult);
                q *= mult;
                break;
            case FLUX_SECRETE:
                flux_secrete_cuda(flux, i, ssi, ssj, &mult);
                q *= mult;
                break;
            case FLUX_UPTAKE:
                flux_uptake_cuda(flux, i, ssi, ssj, &mult);
                q *= mult;
                break;
            default:
                __builtin_unreachable();
        }
        
        float half_decay = flux.decay_coef[i] * 0.5;
        qi[flux.indices_a[i]] -= q + half_decay * ssi;
        qj[flux.indices_b[i]] += q - half_decay * ssj;
    }
}


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__ inline void runner_dosort_cuda(cuda::Particle *parts_i, int count_i, unsigned int *sort_i, int sid) {
    
    TIMER_TIC
    
    /* Get the shift vector from the sid. */
    float shift[3];
    shift[0] = cuda_shift[ 3*sid + 0 ] * hip/hip_runtime.h[0];
    shift[1] = cuda_shift[ 3*sid + 1 ] * hip/hip_runtime.h[1];
    shift[2] = cuda_shift[ 3*sid + 2 ] * hip/hip_runtime.h[2];

    /* Pre-compute the inverse norm of the shift. */
    float nshift = sqrtf(shift[0]*shift[0] + shift[1]*shift[1] + shift[2]*shift[2]);
    shift[0] = cuda_shiftn[ 3*sid + 0 ];
    shift[1] = cuda_shiftn[ 3*sid + 1 ];
    shift[2] = cuda_shiftn[ 3*sid + 2 ];

    /* Pack the parts into the sort arrays. */
    for(int k = threadIdx.x ; k < count_i ; k += blockDim.x) {
        float4 pix = parts_i[ k ].x;
        sort_i[k] =(k << 16) | (unsigned int)(cuda_dscale * (nshift + pix.x*shift[0] + pix.y*shift[1] + pix.z*shift[2]));
    }

    TIMER_TOC(tid_pack)
    __syncthreads();
    /* Sort using normalized bitonic sort. */
    cuda_sort_descending(sort_i, count_i);

}


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__  void runner_dopair_left_cuda(cuda::Particle * parts_i, int count_i, cuda::Particle *parts_j, int count_j, float *forces_i, unsigned int *sort_i, unsigned int *sort_j, float *shift, unsigned int dshift, float *epot_global) {
    float epot = 0.0f;
    
    TIMER_TIC
    
    /* Pre-compute the inverse norm of the shift. */
    unsigned int dmaxdist = 2 + cuda_dscale * cuda_maxdist - dshift;
    
    for(int i = threadIdx.x ; i < count_i ;  i += blockDim.x) {

        unsigned int di = (sort_i[i]&0xffff) + dmaxdist;
        /* Get a direct pointer on the pjdth part in cell_j. */
        int spid = sort_i[i] >> 16;
        cuda::Particle pi = parts_i[ spid ];
        float4 pix = float4(pi.x);
        pix.x -= shift[0]; pix.y -= shift[1]; pix.z -= shift[2];
        float pif[] = {0.0f, 0.0f, 0.0f, 0.0f};

        /* Loop over the particles in cell_j. */
        for(int k = count_j-1 ; k >=0 && (sort_j[k]&0xffff) <= di ; k--) {
                 
            /* Get a handle on the wrapped particle pid in cell_i. */

            cuda::Particle pj = parts_j[ sort_j[k] >> 16 ];

            /* Compute the radius between pi and pj. */
            float3 dx{pix.x - pj.x.x, pix.y - pj.x.y, pix.z - pj.x.z};
            float r2 = dx.x * dx.x + dx.y * dx.y + dx.z * dx.z;

            if(r2 >= cuda_cutoff2) {
                continue;
            }

            float number_density;
            
            w_cubic_spline_cuda_nr(r2, cuda_cutoff, &number_density);

            pif[3] += number_density;

            int pind, nr_pots, nr_dpds;
            bool iscluster = pi.w.w & PARTICLE_BOUND && pj.w.w & PARTICLE_BOUND && pi.w.z == pj.w.z;

            if(iscluster) {
                pind    = cuda_pind_cluster[ ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y)    ];
                nr_pots = cuda_pind_cluster[ ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y) + 1];
                nr_dpds = cuda_pind_cluster[ ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y) + 2];
            }
            else {
                pind    = cuda_pind[ ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y)    ];
                nr_pots = cuda_pind[ ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y) + 1];
                nr_dpds = cuda_pind[ ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y) + 2];
            }

            if(pind == 0) 
                continue;

            float ee = 0.0f;
            float pjf[4];

            /* Interact particles pi and pj. */
            if(iscluster) 
                potential_eval_super_ex_cuda<true>(pind, nr_pots, nr_dpds, pi, pj, dx, r2, &ee, pif, pjf);
            else 
                potential_eval_super_ex_cuda<false>(pind, nr_pots, nr_dpds, pi, pj, dx, r2, &ee, pif, pjf);

            /* Store the interaction energy. */
            epot += ee;

        } /* loop over parts in cell_i. */

        /* Update the force on pj. */
        for(int k = 0 ; k < 4 ; k++)
        	atomicAdd(&forces_i[ 4*spid + k], pif[k]);
        
    } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
    
}


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__  void runner_dostate_left_cuda(cuda::Particle * parts_i, float *states_i, int count_i, cuda::Particle *parts_j, float *states_j, int count_j, float *fluxes_i, unsigned int *sort_i, unsigned int *sort_j, float *shift, unsigned int dshift, unsigned int nr_states) {
    
    TIMER_TIC
    
    /* Pre-compute the inverse norm of the shift. */
    unsigned int dmaxdist = 2 + cuda_dscale * cuda_maxdist - dshift;
    
    for(int i = threadIdx.x ; i < count_i ;  i += blockDim.x) {

        unsigned int di = (sort_i[i]&0xffff) + dmaxdist;
        /* Get a direct pointer on the pjdth part in cell_j. */
        int spid = sort_i[i] >> 16;
        cuda::Particle pi = parts_i[ spid ];
        float4 pix = float4(pi.x);
        pix.x -= shift[0]; pix.y -= shift[1]; pix.z -= shift[2];

        float pifx[TF_SIMD_SIZE];
        memset(pifx, 0.f, nr_states * sizeof(float));
        float *pis = &states_i[nr_states * spid];

        /* Loop over the particles in cell_j. */
        for(int k = count_j-1 ; k >=0 && (sort_j[k]&0xffff) <= di ; k--) {
                 
            /* Get a handle on the wrapped particle pid in cell_i. */

            int spjd = sort_j[k] >> 16;
            cuda::Particle pj = parts_j[ spjd ];

            /* Compute the radius between pi and pj. */
            float r2 = 0.0f, dx;
            dx = pix.x - pj.x.x; r2 += dx * dx;
            dx = pix.y - pj.x.y; r2 += dx * dx;
            dx = pix.z - pj.x.z; r2 += dx * dx;

            if(r2 >= cuda_cutoff2) {
                continue;
            }

            int fxind = cuda_fxind[pi.w.y * cuda_maxtype + pj.w.y];
            if(fxind != 0) {
                flux_eval_ex_cuda(cuda_fluxes[fxind], sqrtf(r2), pis, &states_j[nr_states * spjd], pi.w.y, pj.w.y, pifx);
            }

        } /* loop over parts in cell_i. */

        for(int k = 0; k < nr_states; k++)
            atomicAdd(&fluxes_i[nr_states * spid + k], pifx[k]);
        
    } /* loop over the particles in cell_j. */
        
    TIMER_TOC(tid_pair)
    
}




/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__ void runner_dopair_right_cuda(cuda::Particle *parts_i, int count_i, cuda::Particle *parts_j, int count_j, float *forces_i, unsigned int *sort_i, unsigned int *sort_j, float *shift, unsigned int dshift, float *epot_global) {
    float epot = 0.0f;
    
    TIMER_TIC
    
    /* Pre-compute the inverse norm of the shift. */
    unsigned int dmaxdist = 2 + cuda_dscale * cuda_maxdist - dshift;

    for(int i = threadIdx.x ; i < count_i ;  i += blockDim.x) {

        unsigned int dj = (sort_i[i]&0xffff) - dmaxdist;
        /* Get a direct pointer on the pjdth part in cell_j. */
        int spid = sort_i[i] >> 16;
        cuda::Particle pi = parts_i[ spid ];
        float4 pix = float4(pi.x);
        pix.x += shift[0]; pix.y += shift[1]; pix.z += shift[2];
        float pif[] = {0.0f, 0.0f, 0.0f, 0.0f};
        
        /* Loop over the particles in cell_j. */
        for(int k = 0 ; k < count_j && dj <= (sort_j[k]&0xffff) ; k++) {

            /* Get a handle on the wrapped particle pid in cell_i. */
            cuda::Particle pj = parts_j[ sort_j[k] >> 16 ];

            /* Compute the radius between pi and pj. */
            float3 dx{pix.x - pj.x.x, pix.y - pj.x.y, pix.z - pj.x.z};
            float r2 = dx.x * dx.x + dx.y * dx.y + dx.z * dx.z;
            
            if(r2 >= cuda_cutoff2) {
                continue;
            }

            float number_density;

            w_cubic_spline_cuda_nr(r2, cuda_cutoff, &number_density);

            pif[3] += number_density;

            int pind, nr_pots, nr_dpds;
            bool iscluster = pi.w.w & PARTICLE_BOUND && pj.w.w & PARTICLE_BOUND && pi.w.z == pj.w.z;

            if(iscluster) {
                pind    = cuda_pind_cluster[ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y)    ];
                nr_pots = cuda_pind_cluster[ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y) + 1];
                nr_dpds = cuda_pind_cluster[ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y) + 2];
            }
            else {
                pind    = cuda_pind[ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y)    ];
                nr_pots = cuda_pind[ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y) + 1];
                nr_dpds = cuda_pind[ENGINE_CUDA_PIND_WIDTH * (pi.w.y * cuda_maxtype + pj.w.y) + 2];
            }
            if(pind == 0) 
                continue;

            float ee = 0.0f;
            float pjf[4];

            /* Interact particles pi and pj. */
            if(iscluster) 
                potential_eval_super_ex_cuda<true>(pind, nr_pots, nr_dpds, pi, pj, dx, r2, &ee, pif, pjf);
            else 
                potential_eval_super_ex_cuda<false>(pind, nr_pots, nr_dpds, pi, pj, dx, r2, &ee, pif, pjf);

            /* Store the interaction force and energy. */
            epot += ee;

        } /* loop over parts in cell_i. */

        /* Update the force on pj. */
        for(int k = 0 ; k < 4 ; k++)
        	atomicAdd(&forces_i[ 4*spid + k], pif[k]);
        
    } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
    
}




/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__ void runner_dostate_right_cuda(cuda::Particle *parts_i, float *states_i, int count_i, cuda::Particle *parts_j, float *states_j, int count_j, float *fluxes_i, unsigned int *sort_i, unsigned int *sort_j, float *shift, unsigned int dshift, unsigned int nr_states) {
    
    TIMER_TIC
    
    /* Pre-compute the inverse norm of the shift. */
    unsigned int dmaxdist = 2 + cuda_dscale * cuda_maxdist - dshift;

    for(int i = threadIdx.x ; i < count_i ;  i += blockDim.x) {

        unsigned int dj = (sort_i[i]&0xffff) - dmaxdist;
        /* Get a direct pointer on the pjdth part in cell_j. */
        int spid = sort_i[i] >> 16;
        cuda::Particle pi = parts_i[ spid ];
        float4 pix = float4(pi.x);
        pix.x += shift[0]; pix.y += shift[1]; pix.z += shift[2];

        float pifx[TF_SIMD_SIZE];
        memset(pifx, 0.f, nr_states * sizeof(float));
        float *pis = &states_i[nr_states * spid];
        
        /* Loop over the particles in cell_j. */
        for(int k = 0 ; k < count_j && dj <= (sort_j[k]&0xffff) ; k++) {

            /* Get a handle on the wrapped particle pid in cell_i. */
            int spjd = sort_j[k] >> 16;
            cuda::Particle pj = parts_j[ spjd ];

            /* Compute the radius between pi and pj. */
            float r2 = 0.0f, dx;
            dx = pix.x - pj.x.x; r2 += dx * dx;
            dx = pix.y - pj.x.y; r2 += dx * dx;
            dx = pix.z - pj.x.z; r2 += dx * dx;
            
            if(r2 >= cuda_cutoff2) {
                continue;
            }

            int fxind = cuda_fxind[pi.w.y * cuda_maxtype + pj.w.y];
            if(fxind != 0) { 
                flux_eval_ex_cuda(cuda_fluxes[fxind], sqrtf(r2), pis, &states_j[nr_states * spjd], pi.w.y, pj.w.y, pifx);
            }

        } /* loop over parts in cell_i. */

        for(int k = 0; k < nr_states; k++)
            atomicAdd(&fluxes_i[nr_states * spid + k], pifx[k]);
        
    } /* loop over the particles in cell_j. */

    TIMER_TOC(tid_pair)
    
}

/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 * @param parts Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__ void runner_dopair_self_cuda (cuda::Particle *parts, int count, int cell_id, float *forces, float *epot_global) {
    float epot = 0.0f;
    unsigned int cell_flags = cuda_cflags[cell_id];
    bool boundary = cell_flags & cell_active_any;
    float3 cdims = cuda_cdims[cell_id];
    
    TIMER_TIC

    /* Loop over the particles in the cell, frame-wise. */
    for(int i = threadIdx.x ; i < count ;  i += blockDim.x) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        cuda::Particle pj = parts[ i ];
        int pjoff = pj.w.y * cuda_maxtype;
        float pjf[] = {0.0f, 0.0f, 0.0f, 0.0f};
        bool iscluster_maybe = pj.w.w & PARTICLE_BOUND;

        if(boundary) {
            boundary_eval_cuda(pj, cdims, cell_flags, pjf, &epot);
        }
        
        /* Loop over the particles in cell_i. */
        for(int k = 0 ; k < count ; k++) {
        	if(i != k) {
                /* Get a handle on the wrapped particle k in cell_i. */
                cuda::Particle pi = parts[ k ];

                /* Compute the radius between pi and pj. */
                float3 dx{pi.x.x - pj.x.x, pi.x.y - pj.x.y, pi.x.z - pj.x.z};
                float r2 = dx.x * dx.x + dx.y * dx.y + dx.z * dx.z;

                if(r2 >= cuda_cutoff2) {
                    continue;
                }

                float number_density;

                w_cubic_spline_cuda_nr(r2, cuda_cutoff, &number_density);

                pjf[3] += number_density;

                int pind, nr_pots, nr_dpds;
                bool iscluster = iscluster_maybe && pi.w.w & PARTICLE_BOUND && pi.w.z == pj.w.z;

                if(iscluster) {
                    pind    = cuda_pind_cluster[ENGINE_CUDA_PIND_WIDTH * (pjoff + pi.w.y)    ];
                    nr_pots = cuda_pind_cluster[ENGINE_CUDA_PIND_WIDTH * (pjoff + pi.w.y) + 1];
                    nr_dpds = cuda_pind_cluster[ENGINE_CUDA_PIND_WIDTH * (pjoff + pi.w.y) + 2];
                }
                else {
                    pind    = cuda_pind[ENGINE_CUDA_PIND_WIDTH * (pjoff + pi.w.y)    ];
                    nr_pots = cuda_pind[ENGINE_CUDA_PIND_WIDTH * (pjoff + pi.w.y) + 1];
                    nr_dpds = cuda_pind[ENGINE_CUDA_PIND_WIDTH * (pjoff + pi.w.y) + 2];
                }
                if(pind == 0) 
                    continue;

                float ee = 0.0f;
                float pif[4];

                /* Interact particles pi and pj. */
                if(iscluster) 
                    potential_eval_super_ex_cuda<true>(pind, nr_pots, nr_dpds, pi, pj, dx, r2, &ee, pif, pjf);
                else 
                    potential_eval_super_ex_cuda<false>(pind, nr_pots, nr_dpds, pi, pj, dx, r2, &ee, pif, pjf);

                /* Store the interaction force and energy. */
                epot += ee;

			}
        } /* loop over parts in cell_i. */

        /* Update the force on pj. */
        for(int k = 0 ; k < 4 ; k++)
        	atomicAdd(&forces[ 4*i + k], pjf[k]);

    } /* loop over the particles in cell_j. */

    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_self)
    
}

/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 * @param parts Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__ void runner_dostate_self_cuda (cuda::Particle *parts, float *states, int count, float *fluxes, unsigned int nr_states) {
    
    TIMER_TIC

    /* Loop over the particles in the cell, frame-wise. */
    for(int i = threadIdx.x ; i < count ;  i += blockDim.x) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        cuda::Particle pj = parts[ i ];
        int pjoff = pj.w.y * cuda_maxtype;

        float pjfx[TF_SIMD_SIZE];
        memset(pjfx, 0.f, nr_states * sizeof(float));
        float *pjs = &states[nr_states * i];
        
        /* Loop over the particles in cell_i. */
        for(int k = 0 ; k < count ; k++) {
        	if(i != k) {
                /* Get a handle on the wrapped particle k in cell_i. */
                cuda::Particle pi = parts[ k ];

                /* Compute the radius between pi and pj. */
                float r2 = 0.0f, dx;
                dx = pi.x.x - pj.x.x; r2 += dx * dx;
                dx = pi.x.y - pj.x.y; r2 += dx * dx;
                dx = pi.x.z - pj.x.z; r2 += dx * dx;

                if(r2 >= cuda_cutoff2) {
                    continue;
                }

                int fxind = cuda_fxind[pjoff + pi.w.y];
                if(fxind != 0) {
                    flux_eval_ex_cuda(cuda_fluxes[fxind], sqrtf(r2), pjs, &states[nr_states * k], pj.w.y, pi.w.y, pjfx);
                }

			}
        } /* loop over parts in cell_i. */

        for(int k = 0; k < nr_states; k++)
            atomicAdd(&fluxes[nr_states * i + k], pjfx[k]);

    } /* loop over the particles in cell_j. */
        
    TIMER_TOC(tid_self)
    
}



/**
 * @brief Our very own memset for the particle forces as hipMemsetAsync requires
 *        a device switch when using streams on different devices.
 *
 */
 
__global__ void cuda_memset_float(float *data, float val, int N) {

    int k, tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    for(k = tid ; k < N ; k += stride)
        data[k] = val;

    }


/**
 * @brief Loop over the cell pairs and process them.
 *
 */
template<bool is_stateful> 
__global__ void runner_run_cuda(float *forces, float *fluxes, int *counts, int *ind, int verlet_rebuild, unsigned int nr_states, unsigned int cuda_nrparts) {
    
    int threadID;
    float epot = 0.0f;
    volatile __shared__ int tid;
    __shared__ float shift[3];
    __shared__ unsigned int dshift;
    extern __shared__ unsigned int sort_arrs[];
    unsigned int *sort_i = (unsigned int*)&sort_arrs[0];
    unsigned int *sort_j = (unsigned int*)&sort_arrs[cuda_nrparts];
    __shared__ float shiftn[39];

	TIMER_TIC2
    
    /* Get the block and thread ids. */
    threadID = threadIdx.x;

    // Copy shifts to shared memory
    for(int i = threadID; i < 39; i += blockDim.x) {
        shiftn[i] = cuda_shiftn[i];
    }

    __syncthreads();

    /* Main loop... */
    while(1) {
    	
    	if(threadID == 0) {
            TIMER_TIC
            tid = runner_cuda_gettask_nolock(&cuda_queues[0], 0);
            TIMER_TOC(tid_gettask)
        }
        
        /*Everyone wait for us to get a task id*/
        __syncthreads();
        
        /* Exit if we didn't get a valid task. */
        
        if(tid < 0) 
            break;
	
        /* Switch task type. */
        if(cuda_tasks[tid].type == task_type_pair) {
	        
            /* Get a hold of the pair cells. */
            
            int cid = cuda_tasks[tid].i;
            int cjd = cuda_tasks[tid].j;
            /*Left interaction*/
            /* Get the shift and dshift vector for this pair. */
            if(threadID == 0) {
                #ifdef TASK_TIMERS
                NAMD_timers[tid].x = blockIdx.x;
                NAMD_timers[tid].y = task_type_pair;
                NAMD_timers[tid].z = clock();
                #endif
                for(int k = 0 ; k < 3 ; k++) {
                    shift[k] = cuda_corig[ 3*cjd + k ] - cuda_corig[ 3*cid + k ];
                    if(2*shift[k] > cuda_dim[k])
                        shift[k] -= cuda_dim[k];
                    else if(2*shift[k] < -cuda_dim[k])
                        shift[k] += cuda_dim[k];
                }
                dshift = cuda_dscale *(shift[0]*shiftn[ 3*cuda_tasks[tid].flags     ] +
                                         shift[1]*shiftn[ 3*cuda_tasks[tid].flags + 1 ] +
                                         shift[2]*shiftn[ 3*cuda_tasks[tid].flags + 2 ]);
            }
            
            /* Load the sorted indices. */

            cuda_memcpy(sort_i, &cuda_sortlists[ 13*ind[cid] + counts[cid]*cuda_tasks[tid].flags ], sizeof(int)*counts[cid]);
            cuda_memcpy(sort_j, &cuda_sortlists[ 13*ind[cjd] + counts[cjd]*cuda_tasks[tid].flags ], sizeof(int)*counts[cjd]);
            __syncthreads();
            
            /* Copy the particle data into the local buffers. */
            cuda::Particle *parts_i = &cuda_parts[ ind[cid] ];
            cuda::Particle *parts_j = &cuda_parts[ ind[cjd] ];
            
            /* Put a finger on the forces. */
            float *forces_i = &forces[ 4*ind[cid] ];
            
            /*Set to left interaction*/
            /* Compute the cell pair interactions. */
            runner_dopair_left_cuda(
                parts_i, counts[cid],
                parts_j, counts[cjd],
                forces_i, 
                sort_i, sort_j,
                shift, dshift, 
                &epot 
            );
            
            float *forces_j = &forces[ 4*ind[cjd] ];

            /*Set to right interaction*/
            /* Compute the cell pair interactions. */
            runner_dopair_right_cuda(
                parts_j, counts[cjd],
                parts_i, counts[cid],
                forces_j, 
                sort_j, sort_i,
                shift, dshift, 
                &epot
            );

            if(is_stateful) {
                
                float *states_i = &cuda_part_states[nr_states * ind[cid]];
                float *states_j = &cuda_part_states[nr_states * ind[cjd]];
                float *fluxes_i = &fluxes[nr_states * ind[cid]];
                
                /*Set to left interaction*/
                /* Compute the cell pair interactions. */
                runner_dostate_left_cuda(
                    parts_i, states_i, counts[cid],
                    parts_j, states_j, counts[cjd],
                    fluxes_i, 
                    sort_i, sort_j,
                    shift, dshift, nr_states
                );
                
                float *fluxes_j = &fluxes[nr_states * ind[cjd]];

                /*Set to right interaction*/
                /* Compute the cell pair interactions. */
                runner_dostate_right_cuda(
                    parts_j, states_j, counts[cjd],
                    parts_i, states_i, counts[cid],
                    fluxes_j, 
                    sort_j, sort_i,
                    shift, dshift, nr_states
                );
            }

            #ifdef TASK_TIMERS
            if(threadID==0)
                NAMD_timers[tid].w = clock();
    	    #endif
            __syncthreads();                    
        }
        else if(cuda_tasks[tid].type == task_type_self) {
        
            #ifdef TASK_TIMERS
            if(threadID==0){
                NAMD_timers[tid].x = blockIdx.x;
                NAMD_timers[tid].y = task_type_self;
                NAMD_timers[tid].z = clock();
            }
    	    #endif
            /* Get a hold of the cell id. */
            int cid = cuda_tasks[tid].i;
            
            /* Put a finger on the forces. */
            float *forces_i = &forces[ 4*ind[cid] ];
                
            /* Copy the particle data into the local buffers. */
            cuda::Particle *parts_j = &cuda_parts[ ind[cid] ];
                
            /* Compute the cell self interactions. */
            runner_dopair_self_cuda(parts_j, counts[cid], cid, forces_i, &epot);

            if(is_stateful) {
                float *fluxes_i = &fluxes[nr_states * ind[cid]];
                float *states_j = &cuda_part_states[nr_states * ind[cid]];
                runner_dostate_self_cuda(parts_j, states_j, counts[cid], fluxes_i, nr_states);
            }

            #ifdef TASK_TIMERS
            if(threadID==0)
            	NAMD_timers[tid].w = clock();
    	    #endif
            __syncthreads();
        }
            
        /* Only do sorts if we have to re-build the pseudo-verlet lists. */
        else if(/*0 &&*/ cuda_tasks[tid].type == task_type_sort && verlet_rebuild) {
        	#ifdef TASK_TIMERS
	        if(threadID==0){
                NAMD_timers[tid].x = blockIdx.x;
				NAMD_timers[tid].y = task_type_sort;
            	NAMD_timers[tid].z = clock();
		    }
    	    #endif
            
            /* Loop over the different sort IDs. */
            if(cuda_tasks[tid].flags != 0) {
                /* Get a hold of the cell id. */
                int cid = cuda_tasks[tid].i;
                
                /* Copy the particle data into the local buffers. */
                cuda::Particle *parts_j = &cuda_parts[ ind[cid] ];

                for(int sid = 0 ; sid < 13 ; sid++) {
                        
                    /* Call the sorting function with the buffer. */
                    runner_dosort_cuda(parts_j, counts[cid], sort_i, sid);
                    __syncthreads();
                    /* Copy the local shared memory back to the global memory. */
                    
                    cuda_memcpy(&cuda_sortlists[ 13*ind[cid] + sid*counts[cid] ], sort_i, sizeof(unsigned int) * counts[cid]);
                    
                    __syncthreads();
            
                }
            }
            #ifdef TASK_TIMERS
            if(threadID==0)
                NAMD_timers[tid].w = clock();
    	    #endif
        		
        }

        /* Unlock any follow-up tasks. */
        if(threadID == 0)
            for(int k = 0 ; k < cuda_tasks[tid].nr_unlock ; k++)
                atomicSub((int *)&cuda_tasks[ cuda_tasks[tid].unlock[k] ].wait, 1);
        
    } /* main loop. */
        
    /* Accumulate the potential energy. */
    epot = epot * 0.5f ;
	/* Accumulate the potential energy. */
    atomicAdd(&cuda_epot, epot);

    /* Make a notch on the barrier, last one out cleans up the mess... */

	if(threadID == 0)
		tid =(atomicAdd(&cuda_barrier, 1) == gridDim.x-1);
	__syncthreads();
    if(tid) {
	    TIMER_TIC

    	if(threadID == 0) {
            cuda_barrier = 0;
            cuda_epot_out = cuda_epot;
            cuda_epot = 0.0f;
            volatile int *temp = cuda_queues[0].data; cuda_queues[0].data = cuda_queues[0].rec_data; cuda_queues[0].rec_data = temp;
            cuda_queues[0].first = 0;
            cuda_queues[0].last = cuda_queues[0].count;
            cuda_queues[0].rec_count = 0;
	        // printf("%i \n", cuda_maxtype);
        }
        // NAMD_barrier=0;
      	for(int j = threadID ; j < cuda_nr_tasks /*myq->count*/ ; j+= blockDim.x)
            for(int k = 0 ; k < cuda_tasks[j].nr_unlock ; k++)
                atomicAdd((int *) &cuda_tasks[ cuda_tasks[j].unlock[k] ].wait, 1);

	    TIMER_TOC(tid_cleanup)
    }
    
    TIMER_TOC2(tid_total)

}


/**
 * @brief Offload and compute the nonbonded interactions on a CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
extern "C" int cuda::engine_nonbond_cuda(struct engine *e) {

    int k, cid, did, pid, fid, maxcount = 0;
    hipStream_t stream;
    hipEvent_t tic, toc_load, toc_run, toc_unload;
    float ms_load, ms_run, ms_unload;
    struct Particle *p;
    cuda::Particle *parts_cuda = (cuda::Particle*)e->parts_cuda_local, *buff_part;
    float *part_states_cuda = (float *)e->part_states_cuda_local, *buff_part_states;
    struct space *s = &e->s;
    FPTYPE maxdist = s->cutoff + 2*s->maxdx;
    int *counts = e->counts_cuda_local[ 0 ], *inds = e->ind_cuda_local[ 0 ];
    float *forces_cuda[ engine_maxgpu ], *fluxes_next_cuda[engine_maxgpu], epot[ engine_maxgpu ];
    unsigned int nr_states = e->nr_fluxes_cuda - 1;
    #ifdef TIMERS
        float timers[ tid_count ];
        double icpms = 1000.0 / 1.4e9; 
    #endif
    
    /* Create the events. */
    if(hipSetDevice(e->devices[e->nr_devices-1]) ||
         hipEventCreate(&tic) != hipSuccess ||
         hipEventCreate(&toc_load) != hipSuccess ||
         hipEventCreate(&toc_run) != hipSuccess ||
         hipEventCreate(&toc_unload) != hipSuccess)
        return cuda_error(engine_err_cuda);
    
    /* Start the clock on the first stream. */
    cuda_safe_call(hipEventRecord(tic, (hipStream_t)e->streams[e->nr_devices-1]));
    
    /* Re-set timers */
    #ifdef TIMERS
        for(int k = 0 ; k < tid_count ; k++)
            timers[k] = 0.0f;
        for(did = 0 ; did < e->nr_devices ; did++)
            cuda_safe_call(hipMemcpyToSymbolAsync(HIP_SYMBOL(cuda_timers), timers, sizeof(float) * tid_count, 0, hipMemcpyHostToDevice, (hipStream_t)e->streams[did]));
    #endif

    std::vector<TissueForge::Particle*> cell_parts;
    cell_parts.reserve(s->nr_cells);
    for(int k = 0; k < s->nr_cells; k++) {
        cell_parts.push_back(s->cells[k].parts);
    }
    
    /* Loop over the devices and call the different kernels on each stream. */
    for(did = 0 ; did < e->nr_devices ; did++) {
    
        /* Set the device ID. */
        cuda_safe_call(hipSetDevice(e->devices[did]));

        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];
        
        /* Load the particle data onto the device. */
        
        counts = e->counts_cuda_local[ did ];
        inds = e->ind_cuda_local[ did ];
        /* Clear the counts array. */
        bzero(counts, sizeof(int) * s->nr_cells);

        /* Load the counts. */
        for(maxcount = 0, k = 0; k < e->cells_cuda_nr[did] ; k++)
        if((counts[e->cells_cuda_local[did][k]] = s->cells[e->cells_cuda_local[did][k]].count) > maxcount)
            maxcount = counts[ e->cells_cuda_local[did][k]];
    /*    for(maxcount = 0, k = 0 ; k < s->nr_marked ; k++)
            if((counts[ s->cid_marked[k] ] = s->cells[ s->cid_marked[k] ].count) > maxcount)
                maxcount = counts[ s->cid_marked[k] ];*/

        /* Raise maxcount to the next multiple of 32. */
        maxcount =(maxcount + (cuda_frame - 1)) & ~(cuda_frame - 1);
        // printf("engine_cuda_load_parts: maxcount=%i.\n", maxcount);

        /* Compute the indices. */
        inds[0] = 0;
        for(k = 1 ; k < e->cells_cuda_nr[did] ; k++)
            inds[k] = inds[k-1] + counts[k-1];

        auto _cells_cuda_local = e->cells_cuda_local[did];

        if(nr_states > 0) {
            auto func = [nr_states, &_cells_cuda_local, &inds, &counts, &cell_parts, &parts_cuda, &part_states_cuda](int k) -> void {

                /* Get the cell id. */
                auto cid = _cells_cuda_local[k];

                /* Copy the particle data to the device. */
                auto buff_part = (cuda::Particle*)&parts_cuda[ inds[cid] ];
                auto buff_part_states = &part_states_cuda[nr_states * inds[cid]];
                for(int pid = 0 ; pid < counts[cid] ; pid++) {
                    TissueForge::Particle *part = &cell_parts[cid][pid];
                    buff_part[ pid ] = cuda::Particle(part, nr_states);
                    for(int ks = 0; ks < nr_states; ks++) 
                        buff_part_states[nr_states * pid + ks] = part->state_vector->fvec[ks];
                }

            };

            parallel_for(e->cells_cuda_nr[did], func);
        } 
        else {
            auto func = [&_cells_cuda_local, &inds, &counts, &cell_parts, &parts_cuda](int k) -> void {

                /* Get the cell id. */
                auto cid = _cells_cuda_local[k];

                /* Copy the particle data to the device. */
                auto buff_part = (cuda::Particle*)&parts_cuda[ inds[cid] ];
                for(int pid = 0 ; pid < counts[cid] ; pid++) {
                    buff_part[ pid ] = cuda::Particle(&cell_parts[cid][pid]);
                }

            };

            parallel_for(e->cells_cuda_nr[did], func);
        }

	    /* Start by setting the maxdist on the device. */
        cuda_safe_call(hipMemcpyToSymbolAsync(HIP_SYMBOL(cuda_maxdist), &maxdist, sizeof(float), 0, hipMemcpyHostToDevice, stream));

        /* Copy the counts onto the device. */
        cuda_safe_call(hipMemcpyAsync(e->counts_cuda[did], counts, sizeof(int) * s->nr_cells, hipMemcpyHostToDevice, stream));

        /* Copy the inds onto the device. */
        cuda_safe_call(hipMemcpyAsync(e->ind_cuda[did], inds, sizeof(int) * s->nr_cells, hipMemcpyHostToDevice, stream));

        /* Bind the particle positions. */
        cuda_safe_call(hipMemcpyAsync(e->parts_cuda[did], parts_cuda, sizeof(cuda::Particle) * s->nr_parts, hipMemcpyHostToDevice, stream));

        if(nr_states > 0) {
            /* Bind the particle states. */
            cuda_safe_call(hipMemcpyAsync(e->part_states_cuda[did], part_states_cuda, sizeof(float) * s->nr_parts * nr_states, hipMemcpyHostToDevice, stream));
        }

    /* Start the clock. */
    // tic = getticks();
	}
    
    /* Lap the clock on the last stream. */
    cuda_safe_call(hipEventRecord(toc_load, (hipStream_t)e->streams[e->nr_devices-1]));
    

        
        

	/* Loop over the devices and call the different kernels on each stream. */
    for(did = 0 ; did < e->nr_devices ; did++) {

	    /* Set the device ID. */
        cuda_safe_call(hipSetDevice(e->devices[did]));

        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];

        /* Clear the force array. */
        cuda_memset_float <<<8,512,0,stream>>>(e->forces_cuda[did], 0.0f, 4 * s->nr_parts);
        if(nr_states > 0) 
            cuda_memset_float<<<8,512,0,stream>>>(e->fluxes_next_cuda[did], 0.0f, nr_states * s->nr_parts);

        dim3 nr_threads(e->nr_threads[did], 1, 1);
        dim3 nr_blocks(std::min(e->nrtasks_cuda[did], e->nr_blocks[did]), 1, 1);
        
        /* Start the appropriate kernel. */
        if(nr_states > 0) {
            runner_run_cuda<true><<<nr_blocks, nr_threads, 2 * maxcount * sizeof(unsigned int), stream>>>(e->forces_cuda[did], e->fluxes_next_cuda[did], e->counts_cuda[did], e->ind_cuda[did], e->s.verlet_rebuild, nr_states, maxcount);
        }
        else {
            runner_run_cuda<false><<<nr_blocks, nr_threads, 2 * maxcount * sizeof(unsigned int), stream>>>(e->forces_cuda[did], e->fluxes_next_cuda[did], e->counts_cuda[did], e->ind_cuda[did], e->s.verlet_rebuild, nr_states, maxcount);
        }
        cuda_safe_call(hipPeekAtLastError());
    }

    // Initialize the return buffers while waiting
    for(did = 0; did < e->nr_devices ; did ++) {
        if((forces_cuda[did] = (float *)malloc(sizeof(float) * 4 * s->nr_parts)) == NULL)
            return error(engine_err_malloc);
        if(nr_states > 0) 
            if((fluxes_next_cuda[did] = (float *)malloc(sizeof(float) * nr_states * s->nr_parts)) == NULL)
                return error(engine_err_malloc);
    }

	for(did = 0; did < e->nr_devices ; did ++) {
	
	    /* Set the device ID. */
        cuda_safe_call(hipSetDevice(e->devices[did]));

        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];
        
        /* Get the forces from the device. */
        cuda_safe_call(hipMemcpyAsync(forces_cuda[did], e->forces_cuda[did], sizeof(float) * 4 * s->nr_parts, hipMemcpyDeviceToHost, stream));

        /* Get the potential energy. */
        cuda_safe_call(hipMemcpyFromSymbolAsync(&epot[did], HIP_SYMBOL(cuda_epot_out), sizeof(float), 0, hipMemcpyDeviceToHost, stream));

        if(nr_states > 0) {
            // Get the flux data
            cuda_safe_call(hipMemcpyAsync(fluxes_next_cuda[did], e->fluxes_next_cuda[did], sizeof(float) * nr_states * s->nr_parts, hipMemcpyDeviceToHost, stream));
        }
        
    }

    std::vector<int> cell_counts;
    cell_counts.reserve(s->nr_cells);
    for(int k = 0; k < s->nr_cells; k++) {
        cell_counts.push_back(s->cells[k].count);
    }
    
    /* Lap the clock on the last stream. */
    cuda_safe_call(hipEventRecord(toc_run, (hipStream_t)e->streams[e->nr_devices-1]));
    
    /* Get and dump timers. */
    #ifdef TIMERS
        cuda_safe_call(hipMemcpyFromSymbolAsync(timers, HIP_SYMBOL(cuda_timers), sizeof(float) * tid_count, 0, hipMemcpyDeviceToHost, (hipStream_t)e->streams[0]));
        printf("engine_nonbond_cuda: timers = [ %.2f ", icpms * timers[0]);
        for(int k = 1 ; k < tid_count ; k++)
            printf("%.2f ", icpms * timers[k]);
        printf("] ms\n");
    #endif

    #ifdef TASK_TIMERS
		int4 NAMD_timers_local[26*cuda_maxcells*3];
		cuda_safe_call(hipMemcpyFromSymbol(NAMD_timers_local, HIP_SYMBOL(NAMD_timers), sizeof(int4)*26*cuda_maxcells*3, 0, hipMemcpyDeviceToHost));
		for(int i = 0; i < e->s.nr_tasks ; i++)
		printf("Task: %i %i %i %i\n", NAMD_timers_local[i].x, NAMD_timers_local[i].y, NAMD_timers_local[i].z, NAMD_timers_local[i].w);

    #endif
    
    /* Check for any missed CUDA errors. */
    cuda_safe_call(hipPeekAtLastError());
        

    /* Loop over the devices. */
    for(did = 0 ; did < e->nr_devices ; did++) {
    
        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];

        /* Wait for the chickens to come home to roost. */
        cuda_safe_call(hipStreamSynchronize(stream));
    
        /* Get the potential energy. */
        e->s.epot += epot[did];

        auto _cells_cuda_local = e->cells_cuda_local[did];
        auto _forces_cuda = forces_cuda[did];
        auto _ind_cuda_local = e->ind_cuda_local[did];

        if(nr_states > 0) {
            auto _fluxes_next_cuda = fluxes_next_cuda[did];

            auto func = [&_cells_cuda_local, &_forces_cuda, &_ind_cuda_local, nr_states, &_fluxes_next_cuda, &cell_counts, &cell_parts](int k) -> void {

                /* Get the cell id. */
                int cid = _cells_cuda_local[k];

                /* Copy the particle data from the device. */
                auto buff_force = &_forces_cuda[ 4*_ind_cuda_local[cid] ];
                auto buff_flux = &_fluxes_next_cuda[nr_states * _ind_cuda_local[cid] ];

                for(int pid = 0 ; pid < cell_counts[cid] ; pid++) {
                    auto p = &cell_parts[cid][pid];
                    p->f[0] += buff_force[ 4*pid ];
                    p->f[1] += buff_force[ 4*pid + 1 ];
                    p->f[2] += buff_force[ 4*pid + 2 ];
                    p->f[3] += buff_force[ 4*pid + 3 ];

                    for(int fid = 0; fid < nr_states; fid++) 
                        p->state_vector->q[fid] += buff_flux[nr_states * pid + fid];
                }

            };

            parallel_for(e->cells_cuda_nr[did], func);
        } 
        else {
            auto func = [&_cells_cuda_local, &_forces_cuda, &_ind_cuda_local, &cell_counts, &cell_parts](int k) -> void {

                /* Get the cell id. */
                int cid = _cells_cuda_local[k];

                /* Copy the particle data from the device. */
                auto buff_force = &_forces_cuda[ 4*_ind_cuda_local[cid] ];
                for(int pid = 0 ; pid < cell_counts[cid] ; pid++) {
                    auto p = &cell_parts[cid][pid];
                    p->f[0] += buff_force[ 4*pid ];
                    p->f[1] += buff_force[ 4*pid + 1 ];
                    p->f[2] += buff_force[ 4*pid + 2 ];
                    p->f[3] += buff_force[ 4*pid + 3 ];
                }

            };

            parallel_for(e->cells_cuda_nr[did], func);
        }

        /* Deallocate the parts array and counts array. */
        free(forces_cuda[did]);
        if(nr_states > 0)
            free(fluxes_next_cuda[did]);
        
    }
        
    /* Check for any missed CUDA errors. */
    cuda_safe_call(hipPeekAtLastError());

    /* Stop the clock on the last stream. */
    if(hipEventRecord(toc_unload, (hipStream_t)e->streams[e->nr_devices-1]) != hipSuccess ||
         hipStreamSynchronize((hipStream_t)e->streams[e->nr_devices-1]) != hipSuccess)
        return cuda_error(engine_err_cuda);
    
    /* Check for any missed CUDA errors. */
    cuda_safe_call(hipPeekAtLastError());
        
    /* Store the timers. */
    if(hipEventElapsedTime(&ms_load, tic, toc_load) != hipSuccess ||
         hipEventElapsedTime(&ms_run, toc_load, toc_run) != hipSuccess ||
         hipEventElapsedTime(&ms_unload, toc_run, toc_unload) != hipSuccess)
        return cuda_error(engine_err_cuda);
    e->timers[ engine_timer_cuda_load ] += ms_load / 1000 * CPU_TPS;
    e->timers[ engine_timer_cuda_dopairs ] += ms_run / 1000 * CPU_TPS;
    e->timers[ engine_timer_cuda_unload ] += ms_unload / 1000 * CPU_TPS;
    
    /* Go away. */
    return engine_err_ok;
    
}



/**
 * @brief Load the cell data onto the CUDA device.
 *
 * @param e The #engine.
 *
 * @return The maximum number of parts per cell or < 0
 *      on error (see #engine_err).
 */
 
extern "C" int cuda::engine_cuda_load_parts(struct engine *e) {
    
    int k, did, cid, pid, maxcount = 0;
    cuda::Particle *parts_cuda = (cuda::Particle*)e->parts_cuda_local;
    struct space *s = &e->s;
    FPTYPE maxdist = s->cutoff + 2*s->maxdx;
    int *counts = e->counts_cuda_local[0], *inds = e->ind_cuda_local[0];
    hipStream_t stream;
    
    /* Clear the counts array. */
    bzero(counts, sizeof(int) * s->nr_cells);

    /* Load the counts. */
    for(maxcount = 0, k = 0 ; k < s->nr_marked ; k++)
        if((counts[ s->cid_marked[k] ] = s->cells[ s->cid_marked[k] ].count) > maxcount)
            maxcount = counts[ s->cid_marked[k] ];

    /* Raise maxcount to the next multiple of 32. */
    maxcount =(maxcount + (cuda_frame - 1)) & ~(cuda_frame - 1);
    // printf("engine_cuda_load_parts: maxcount=%i.\n", maxcount);

    /* Compute the indices. */
    inds[0] = 0;
    for(k = 1 ; k < s->nr_cells ; k++)
        inds[k] = inds[k-1] + counts[k-1];

    /* Loop over the marked cells. */
    auto func_alloc_marked = [&s, &parts_cuda, &inds, &counts] (int _k) -> void {

        /* Get the cell id. */
        int _cid = s->cid_marked[_k];

        /* Copy the particle data to the device. */
        cuda::Particle *buff = (cuda::Particle*)&parts_cuda[ inds[_cid] ];
        for(int _pid = 0 ; _pid < counts[_cid] ; _pid++) {
            buff[ _pid ] = cuda::Particle(&s->cells[_cid].parts[_pid]);
        }

    };
    parallel_for(s->nr_marked, func_alloc_marked);

    // printf("engine_cuda_load_parts: packed %i cells with %i parts each (%i kB).\n", s->nr_cells, maxcount, (sizeof(float4)*maxcount*s->nr_cells)/1024);

    /* Loop over the devices. */
    for(did = 0 ; did < e->nr_devices ; did++) {
    
        /* Set the device ID. */
        cuda_safe_call(hipSetDevice(e->devices[did]));

        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];
        
        /* Start by setting the maxdist on the device. */
        cuda_safe_call(hipMemcpyToSymbolAsync(HIP_SYMBOL(cuda_maxdist), &maxdist, sizeof(float), 0, hipMemcpyHostToDevice, stream));

        /* Copy the counts onto the device. */
        cuda_safe_call(hipMemcpyAsync(e->counts_cuda[did], counts, sizeof(int) * s->nr_cells, hipMemcpyHostToDevice, stream));

        /* Copy the inds onto the device. */
        cuda_safe_call(hipMemcpyAsync(e->ind_cuda[did], inds, sizeof(int) * s->nr_cells, hipMemcpyHostToDevice, stream));

        /* Bind the particle positions. */
        cuda_safe_call(hipMemcpyAsync(e->parts_cuda[did], parts_cuda, sizeof(cuda::Particle) * s->nr_parts, hipMemcpyHostToDevice, stream));

        /* Clear the force array. */
        cuda_safe_call(hipMemsetAsync(e->forces_cuda[did], 0, sizeof(float) * 4 * s->nr_parts, stream));

    }
    
    /* Our work is done here. */
    return maxcount;

}
    
    

/**
 * @brief Load the cell data from the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
extern "C" int cuda::engine_cuda_unload_parts(struct engine *e) {

    int k, did, cid, pid;
    struct TissueForge::Particle *p;
    float *forces_cuda[ engine_maxgpu ], *buff, epot[ engine_maxgpu ];
    struct space *s = &e->s;
    hipStream_t stream;
    
    /* Loop over the devices. */
    for(did = 0 ; did < e->nr_devices ; did++) {
    
        /* Set the device ID. */
        cuda_safe_call(hipSetDevice(e->devices[did]));

        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];
    
        /* Get the forces from the device. */
        if((forces_cuda[did] = (float *)malloc(sizeof(float) * 4 * s->nr_parts)) == NULL)
            return error(engine_err_malloc);
        cuda_safe_call(hipMemcpyAsync(forces_cuda[did], e->forces_cuda[did], sizeof(float) * 4 * s->nr_parts, hipMemcpyDeviceToHost, stream));

        /* Get the potential energy. */
        cuda_safe_call(hipMemcpyFromSymbolAsync(&epot[did], HIP_SYMBOL(cuda_epot_out), sizeof(float), 0, hipMemcpyDeviceToHost, stream));
        
    }

    /* Loop over the devices. */
    for(did = 0 ; did < e->nr_devices ; did++) {
    
        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];

        /* Wait for the chickens to come home to roost. */
        cuda_safe_call(hipStreamSynchronize(stream));
    
        /* Get the potential energy. */
        e->s.epot += epot[did];
        
        /* Loop over the marked cells. */
        for(k = 0 ; k < s->nr_marked ; k++) {

            /* Get the cell id. */
            cid = s->cid_marked[k];

            /* Copy the particle data from the device. */
            buff = &forces_cuda[did][ 4*e->ind_cuda_local[did][cid] ];
            for(pid = 0 ; pid < s->cells[cid].count ; pid++) {
                p = &s->cells[cid].parts[pid];
                p->f[0] += buff[ 4*pid ];
                p->f[1] += buff[ 4*pid + 1 ];
                p->f[2] += buff[ 4*pid + 2 ];
                p->f[3] += buff[ 4*pid + 3 ];
                }

            }

        /* Deallocate the parts array and counts array. */
        free(forces_cuda[did]);
        
    }
        
    /* Our work is done here. */
    return engine_err_ok;

}

/**
 * @brief Load the queues onto the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
int engine_cuda_queues_load(struct engine *e) {
    
    int did, nr_queues, qid, k, qsize, nr_tasks = e->s.nr_tasks;
    struct hipDeviceProp_t prop;
    int *data;
    struct cuda::queue_cuda queues[ cuda_maxqueues ];
    
    /* Loop over the devices. */
    for(did = 0 ; did < e->nr_devices ; did++) {
    
        /* Set the device ID. */
        cuda_safe_call(hipSetDevice(e->devices[did]));

        /* Get the device properties. */
        cuda_safe_call(hipGetDeviceProperties(&prop, e->devices[did]));

        /* Get the number of SMs on the current device. */
        nr_queues = 1; // prop.multiProcessorCount;

        /* Get the local number of tasks. */
        nr_tasks = e->nrtasks_cuda[did];

        /* Set the size of each queue. */
        qsize = 3 * nr_tasks / min(nr_queues, e->nr_runners);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_queue_size), &qsize, sizeof(int), 0, hipMemcpyHostToDevice));

        /* Allocate a temporary buffer for the queue data. */
        if((data = (int *)malloc(sizeof(int) * qsize)) == NULL)
            return error(engine_err_malloc);

        /* Set the number of queues. */
        e->nr_queues_cuda = nr_queues;

        /* Init each queue separately. */
        for(qid = 0 ; qid < nr_queues ; qid++) {

            /* Fill the data for this queue. */
            queues[qid].count = 0;
            for(k = qid ; k < nr_tasks ; k += nr_queues)
                data[ queues[qid].count++ ] = k;
            for(k = queues[qid].count ; k < qsize ; k++)
                data[k] = -1;

            /* Allocate and copy the data. */
            cuda_safe_call(hipMalloc(&queues[qid].data, sizeof(int) * qsize));
            cuda_safe_call(hipMemcpy((void *)queues[qid].data, data, sizeof(int) * qsize, hipMemcpyHostToDevice));

            /* Allocate and copy the recycling data. */
            for(k = 0 ; k < queues[qid].count ; k++)
                data[k] = -1;
            
            cuda_safe_call(hipMalloc(&queues[qid].rec_data, sizeof(int) * qsize));
            cuda_safe_call(hipMemcpy((void *)queues[qid].rec_data, data, sizeof(int) * qsize, hipMemcpyHostToDevice));

            /* Set some other values. */
            queues[qid].first = 0;
            queues[qid].last = queues[qid].count;
            queues[qid].rec_count = 0;

        }

        /* Copy the queue structures to the device. */
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_queues), queues, sizeof(struct cuda::queue_cuda) * nr_queues, 0, hipMemcpyHostToDevice));

        /* Wait so that we can re-use the local memory. */            
        cuda_safe_call(hipDeviceSynchronize());

        /* Clean up. */
        free(data);
        
    }
        
    /* Fade to grey. */
    return engine_err_ok;

}


__global__ void engine_cuda_queues_finalize_device(int nr_queues) {
    if(threadIdx.x != 0 || blockIdx.x != 0) {
        return;
    }

    for(int qid = 0; qid < nr_queues; qid++) {

        if(hipFree(&cuda_queues[qid].data) != hipSuccess) {
            printf("%s\n", "engine_cuda_queues_finalize_device failed (data)!");
            return;
        }

        if(hipFree(&cuda_queues[qid].rec_data) != hipSuccess) {
            printf("%s\n", "engine_cuda_queues_finalize_device failed (rec_data)!");
            return;
        }

    }

    if(hipFree(cuda_queues) != hipSuccess) {
        printf("%s\n", "engine_cuda_queues_finalize_device failed (cuda_queues)!");
        return;
    }
}

/**
 * @brief Close the run configuration on the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
int cuda::engine_cuda_queues_finalize(struct engine *e) {

    /* Loop over the devices. */
    for(int did = 0 ; did < e->nr_devices ; did++) {
        
        // Set the device ID
        
        cuda_safe_call(hipSetDevice(e->devices[did]));

        // Free queues
        
        engine_cuda_queues_finalize_device<<<1, 1>>>(e->nr_queues_cuda);

        cuda_safe_call(hipDeviceSynchronize());

    }

    return engine_err_ok;
}

/**
 * @brief Load the potentials onto the CUDA device
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
extern "C" int cuda::engine_cuda_load_pots(struct engine *e) {
    int i, j, nr_pots, nr_pots_cluster;
    int nr_devices = e->nr_devices;
    std::vector<int> pind, pind_cluster;

    // Pack the potentials

    int max_coeffs, max_pots, max_dpds;
    int max_coeffs_cluster, max_pots_cluster, max_dpds_cluster;
    std::vector<float> pot_alpha, pot_alpha_cluster;
    std::vector<float> pot_c, pot_c_cluster;
    std::vector<float> pot_dataf, pot_dataf_cluster;
    std::vector<int> pot_datai, pot_datai_cluster;
    std::vector<float> dpd_cf, dpd_cf_cluster;
    std::vector<float> dpd_dataf, dpd_dataf_cluster;
    std::vector<int> dpd_datai, dpd_datai_cluster;
    cuda_safe_call_e(engine_cuda_build_pots_pack(
        e->p, e->max_type * e->max_type, 
        pind, pot_alpha, pot_c, pot_dataf, pot_datai, 
        dpd_cf, dpd_dataf, dpd_datai, 
        max_coeffs, max_pots, max_dpds, nr_pots), engine_err_ok);
    cuda_safe_call_e(engine_cuda_build_pots_pack(
        e->p_cluster, e->max_type * e->max_type, 
        pind_cluster, pot_alpha_cluster, pot_c_cluster, pot_dataf_cluster, pot_datai_cluster, 
        dpd_cf_cluster, dpd_dataf_cluster, dpd_datai_cluster, 
        max_coeffs_cluster, max_pots_cluster, max_dpds_cluster, nr_pots_cluster), engine_err_ok);
    
    /* Store pind as a constant. */

    for(int did = 0 ; did < e->nr_devices ; did++) {
        cuda_safe_call(hipSetDevice(e->devices[did]));
        cuda_safe_call(hipMalloc(&e->pind_cuda[did], sizeof(int) * pind.size()));
        cuda_safe_call(hipMemcpy(e->pind_cuda[did], pind.data(), sizeof(int) * pind.size(), hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_pind), &e->pind_cuda[did], sizeof(int *), 0, hipMemcpyHostToDevice));
    }
    
    for(int did = 0 ; did < e->nr_devices ; did++) {
        cuda_safe_call(hipSetDevice(e->devices[did]));
        cuda_safe_call(hipMalloc(&e->pind_cluster_cuda[did], sizeof(int) * pind_cluster.size()));
        cuda_safe_call(hipMemcpy(e->pind_cluster_cuda[did], pind_cluster.data(), sizeof(int) * pind_cluster.size(), hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_pind_cluster), &e->pind_cluster_cuda[did], sizeof(int *), 0, hipMemcpyHostToDevice));
    }

    // Store the potentials

    hipChannelFormatDesc channelDesc_int = hipCreateChannelDesc<int>();
    hipChannelFormatDesc channelDesc_float = hipCreateChannelDesc<float>();

    for(int did = 0 ; did < e->nr_devices ; did++) {
        cuda_safe_call(hipSetDevice(e->devices[did]));

        cuda_safe_call(hipMallocArray(&cuda_pot_alpha[did], &channelDesc_float, ENGINE_CUDA_POT_WIDTH_ALPHA, pot_alpha.size() / ENGINE_CUDA_POT_WIDTH_ALPHA, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_pot_alpha[did], 0, 0, pot_alpha.data(), sizeof(float) * pot_alpha.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_alpha[did], cuda_pot_alpha[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_alpha), &tex_pot_alpha[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_pot_cluster_alpha[did], &channelDesc_float, ENGINE_CUDA_POT_WIDTH_ALPHA, pot_alpha_cluster.size() / ENGINE_CUDA_POT_WIDTH_ALPHA, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_pot_cluster_alpha[did], 0, 0, pot_alpha_cluster.data(), sizeof(float) * pot_alpha_cluster.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_cluster_alpha[did], cuda_pot_cluster_alpha[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_cluster_alpha), &tex_pot_cluster_alpha[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_pot_c[did], &channelDesc_float, potential_chunk * max_coeffs, pot_c.size() / (potential_chunk * max_coeffs), 0));
        cuda_safe_call(hipMemcpyToArray(cuda_pot_c[did], 0, 0, pot_c.data(), sizeof(float) * pot_c.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_c[did], cuda_pot_c[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_c), &tex_pot_c[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_pot_cluster_c[did], &channelDesc_float, potential_chunk * max_coeffs_cluster, pot_c_cluster.size() / (potential_chunk * max_coeffs_cluster), 0));
        cuda_safe_call(hipMemcpyToArray(cuda_pot_cluster_c[did], 0, 0, pot_c_cluster.data(), sizeof(float) * pot_c_cluster.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_cluster_c[did], cuda_pot_cluster_c[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_cluster_c), &tex_pot_cluster_c[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_pot_dataf[did], &channelDesc_float, ENGINE_CUDA_POT_WIDTH_DATAF, pot_dataf.size() / ENGINE_CUDA_POT_WIDTH_DATAF, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_pot_dataf[did], 0, 0, pot_dataf.data(), sizeof(float) * pot_dataf.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_dataf[did], cuda_pot_dataf[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_dataf), &tex_pot_dataf[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_pot_cluster_dataf[did], &channelDesc_float, ENGINE_CUDA_POT_WIDTH_DATAF, pot_dataf_cluster.size() / ENGINE_CUDA_POT_WIDTH_DATAF, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_pot_cluster_dataf[did], 0, 0, pot_dataf_cluster.data(), sizeof(float) * pot_dataf_cluster.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_cluster_dataf[did], cuda_pot_cluster_dataf[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_cluster_dataf), &tex_pot_cluster_dataf[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_pot_datai[did], &channelDesc_int, ENGINE_CUDA_POT_WIDTH_DATAI, pot_datai.size() / ENGINE_CUDA_POT_WIDTH_DATAI, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_pot_datai[did], 0, 0, pot_datai.data(), sizeof(int) * pot_datai.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_datai[did], cuda_pot_datai[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_datai), &tex_pot_datai[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_pot_cluster_datai[did], &channelDesc_int, ENGINE_CUDA_POT_WIDTH_DATAI, pot_datai_cluster.size() / ENGINE_CUDA_POT_WIDTH_DATAI, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_pot_cluster_datai[did], 0, 0, pot_datai_cluster.data(), sizeof(int) * pot_datai_cluster.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_pot_cluster_datai[did], cuda_pot_cluster_datai[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_pot_cluster_datai), &tex_pot_cluster_datai[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_dpd_cfs[did], &channelDesc_float, ENGINE_CUDA_DPD_WIDTH_CF, dpd_cf.size() / ENGINE_CUDA_DPD_WIDTH_CF, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_dpd_cfs[did], 0, 0, dpd_cf.data(), sizeof(float) * dpd_cf.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_dpd_cfs[did], cuda_dpd_cfs[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_dpd_cfs), &tex_dpd_cfs[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_dpd_cluster_cfs[did], &channelDesc_float, ENGINE_CUDA_DPD_WIDTH_CF, dpd_cf_cluster.size() / ENGINE_CUDA_DPD_WIDTH_CF, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_dpd_cluster_cfs[did], 0, 0, dpd_cf_cluster.data(), sizeof(float) * dpd_cf_cluster.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_dpd_cluster_cfs[did], cuda_dpd_cluster_cfs[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_dpd_cluster_cfs), &tex_dpd_cluster_cfs[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_dpd_dataf[did], &channelDesc_float, ENGINE_CUDA_DPD_WIDTH_DATAF, dpd_dataf.size() / ENGINE_CUDA_DPD_WIDTH_DATAF, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_dpd_dataf[did], 0, 0, dpd_dataf.data(), sizeof(float) * dpd_dataf.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_dpd_dataf[did], cuda_dpd_dataf[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_dpd_dataf), &tex_dpd_dataf[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_dpd_cluster_dataf[did], &channelDesc_float, ENGINE_CUDA_DPD_WIDTH_DATAF, dpd_dataf_cluster.size() / ENGINE_CUDA_DPD_WIDTH_DATAF, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_dpd_cluster_dataf[did], 0, 0, dpd_dataf_cluster.data(), sizeof(float) * dpd_dataf_cluster.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_dpd_cluster_dataf[did], cuda_dpd_cluster_dataf[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_dpd_cluster_dataf), &tex_dpd_cluster_dataf[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_dpd_datai[did], &channelDesc_int, ENGINE_CUDA_DPD_WIDTH_DATAI, dpd_datai.size() / ENGINE_CUDA_DPD_WIDTH_DATAI, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_dpd_datai[did], 0, 0, dpd_datai.data(), sizeof(int) * dpd_datai.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_dpd_datai[did], cuda_dpd_datai[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_dpd_datai), &tex_dpd_datai[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMallocArray(&cuda_dpd_cluster_datai[did], &channelDesc_int, ENGINE_CUDA_DPD_WIDTH_DATAI, dpd_datai_cluster.size() / ENGINE_CUDA_DPD_WIDTH_DATAI, 0));
        cuda_safe_call(hipMemcpyToArray(cuda_dpd_cluster_datai[did], 0, 0, dpd_datai_cluster.data(), sizeof(int) * dpd_datai_cluster.size(), hipMemcpyHostToDevice));
        cuda_safe_call_e(engine_cuda_texture_init(&tex_dpd_cluster_datai[did], cuda_dpd_cluster_datai[did]), engine_err_ok);
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tex_dpd_cluster_datai), &tex_dpd_cluster_datai[did], sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_pots_max), &max_pots, sizeof(int), 0, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_pots_cluster_max), &max_pots_cluster, sizeof(int), 0, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_dpds_max), &max_dpds, sizeof(int), 0, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_dpds_cluster_max), &max_dpds_cluster, sizeof(int), 0, hipMemcpyHostToDevice));
    }
    
    e->nr_pots_cuda = nr_pots;
    e->nr_pots_cluster_cuda = nr_pots_cluster;

    return engine_err_ok;
}

/**
 * @brief Unload the potentials on the CUDA device
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
extern "C" int cuda::engine_cuda_unload_pots(struct engine *e) {

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call(hipSetDevice(e->devices[did]));

        // Free the potentials.

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_alpha[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_pot_alpha[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_cluster_alpha[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_pot_cluster_alpha[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_c[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_pot_c[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_cluster_c[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_pot_cluster_c[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_dataf[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_pot_dataf[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_cluster_dataf[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_pot_cluster_dataf[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_datai[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_pot_datai[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_pot_cluster_datai[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_pot_cluster_datai[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_dpd_cfs[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_dpd_cfs[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_dpd_cluster_cfs[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_dpd_cluster_cfs[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_dpd_dataf[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_dpd_dataf[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_dpd_cluster_dataf[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_dpd_cluster_dataf[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_dpd_datai[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_dpd_datai[did]));

        cuda_safe_call_e(engine_cuda_texture_finalize(tex_dpd_cluster_datai[did]), engine_err_ok);
        cuda_safe_call(hipFreeArray(cuda_dpd_cluster_datai[did]));
        
        cuda_safe_call(::hipFree(e->pind_cuda[did]));
        
        cuda_safe_call(::hipFree(e->pind_cluster_cuda[did]));

    }

    e->nr_pots_cuda = 0;
    e->nr_pots_cluster_cuda = 0;

    return engine_err_ok;
}

/**
 * @brief Refresh the potentials on the CUDA device. 
 * 
 * Can be safely called while on the CUDA device to reload all potential data from the engine. 
 * 
 * @param e The #engine
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err)
 */
extern "C" int cuda::engine_cuda_refresh_pots(struct engine *e) {
    
    if(engine_cuda_unload_pots(e) < 0)
        return error(engine_err);

    if(engine_cuda_load_pots(e) < 0)
        return error(engine_err);

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call(hipSetDevice(e->devices[did]));

        cuda_safe_call(hipDeviceSynchronize());

    }

    return engine_err_ok;
}


/**
 * @brief Allocates particle buffers. Must be called before running on a CUDA device. 
 * 
 * @param e The #engine
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
int engine_cuda_allocate_particles(struct engine *e) {

    /* Allocate the particle buffer. */
    if((e->parts_cuda_local = (cuda::Particle*)malloc(sizeof(cuda::Particle) * e->s.size_parts)) == NULL)
        return error(engine_err_malloc);

    /* Allocate the particle and force data. */
    for(int did = 0; did < e->nr_devices; did++) {
        cuda_safe_call (hipSetDevice(e->devices[did]));
        cuda_safe_call (hipMalloc(&e->parts_cuda[did], sizeof(cuda::Particle) * e->s.size_parts));
        cuda_safe_call (hipMemcpyToSymbol(HIP_SYMBOL(cuda_parts), &e->parts_cuda[did], sizeof(cuda::Particle*), 0, hipMemcpyHostToDevice));
        cuda_safe_call (hipMalloc(&e->forces_cuda[did], sizeof(float) * 4 * e->s.size_parts));
    }

    return engine_err_ok;
}


/**
 * @brief Closes particle buffers. 
 * 
 * @param e The #engine
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
int engine_cuda_finalize_particles(struct engine *e) {

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call(hipSetDevice(e->devices[did]));

        // Free the particle and force data

        cuda_safe_call(hipFree(e->parts_cuda[did]));

        cuda_safe_call(hipFree(e->forces_cuda[did]));

    }

    // Free the particle buffer

    free(e->parts_cuda_local);

    return engine_err_ok;
}


/**
 * @brief Refreshes particle buffers. Can be safely used to resize buffers while running on CUDA device. 
 * 
 * @param e The #engine
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
extern "C" int cuda::engine_cuda_refresh_particles(struct engine *e) {

    if(engine_cuda_finalize_particles(e) < 0)
        return cuda_error(engine_err_cuda);

    if(engine_cuda_allocate_particles(e) < 0)
        return cuda_error(engine_err_cuda);

    bool is_stateful = e->nr_fluxes_cuda > 1;

    if(is_stateful && engine_cuda_refresh_particle_states(e) < 0) 
        return error(engine_err_cuda);

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call(hipSetDevice(e->devices[did]));

        cuda_safe_call(hipDeviceSynchronize());

    }

    return engine_err_ok;
}

extern "C" int cuda::engine_cuda_allocate_particle_states(struct engine *e) {
    int nr_states = e->nr_fluxes_cuda - 1;

    if(nr_states <= 0 || nr_states == engine_cuda_nr_states) 
        return engine_err_ok;

    if(engine_cuda_nr_states > 0) 
        if(engine_cuda_finalize_particle_states(e) < 0) 
            return error(engine_err);
    
    if((e->part_states_cuda_local = (float*)malloc(sizeof(float) * e->s.size_parts * nr_states)) == NULL)
        return error(engine_err_malloc);

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call (hipMalloc(&e->part_states_cuda[did], sizeof(float) * e->s.size_parts * nr_states));
        
        cuda_safe_call (hipMemcpyToSymbol(HIP_SYMBOL(cuda_part_states), &e->part_states_cuda[did], sizeof(float *), 0, hipMemcpyHostToDevice));

    }

    engine_cuda_nr_states = nr_states;

    return engine_err_ok;
}

extern "C" int cuda::engine_cuda_finalize_particle_states(struct engine *e) {
    if(engine_cuda_nr_states == 0) 
        return engine_err_ok;

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call(hipSetDevice(e->devices[did]));

        cuda_safe_call(::hipFree(e->part_states_cuda[did]));

    }

    // Free the particle buffer

    free(e->part_states_cuda_local);

    engine_cuda_nr_states = 0;

    return engine_err_ok;
}

extern "C" int cuda::engine_cuda_refresh_particle_states(struct engine *e) {
    if(engine_cuda_finalize_particle_states(e) < 0)
        return cuda_error(engine_err_cuda);

    if(engine_cuda_allocate_particle_states(e) < 0)
        return cuda_error(engine_err_cuda);

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call(hipSetDevice(e->devices[did]));

        cuda_safe_call(hipDeviceSynchronize());

    }

    return engine_err_ok;
}


/**
 * @brief Load the potentials and cell pairs onto the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
extern "C" int cuda::engine_cuda_load(struct engine *e) {

    int i, k, nr_tasks, c1,c2;
    int did, *cellsorts;
    struct space *s = &e->s;
    int nr_devices = e->nr_devices;
    struct task_cuda *tasks_cuda, *tc, *ts;
    struct task *t;
    float dt = e->dt, cutoff = e->s.cutoff, cutoff2 = e->s.cutoff2, dscale; //, buff[ e->nr_types ];
    float h[3], dim[3], *corig;
    void *dummy[ engine_maxgpu ];
    unsigned int *cflags;
    float3 *cdims;

    /*Split the space over the available GPUs*/
    engine_split_gpu(e, nr_devices, engine_split_GPU);
    
    /* Copy the cell edge lengths to the device. */
    h[0] = s->h[0]*s->span[0];
    h[1] = s->h[1]*s->span[1];
    h[2] = s->h[2]*s->span[2];
    dim[0] = s->dim[0]; dim[1] = s->dim[1]; dim[2] = s->dim[2];
    for(did = 0 ; did < nr_devices ; did++) {
        cuda_safe_call(hipSetDevice(e->devices[did]));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_h), h, sizeof(float) * 3, 0, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_dim), dim, sizeof(float) * 3, 0, hipMemcpyHostToDevice));
    }

    /* Copy the cell origins, dimensions and flags to the device. */
    if((corig = (float *)malloc(sizeof(float) * s->nr_cells * 3)) == NULL)
        return error(engine_err_malloc);
    if((cdims = (float3*)malloc(sizeof(float3) * s->nr_cells)) == NULL)
        return error(engine_err_malloc);
    if((cflags = (unsigned int*)malloc(sizeof(unsigned int) * s->nr_cells)) == NULL)
        return error(engine_err_malloc);
    
    auto func_copy_cell_data = [&s, &corig, &cdims, &cflags](int _i) -> void {
        corig[3*_i + 0] = s->cells[_i].origin[0];
        corig[3*_i + 1] = s->cells[_i].origin[1];
        corig[3*_i + 2] = s->cells[_i].origin[2];
        cdims[_i] = make_float3(s->cells[_i].dim[0], s->cells[_i].dim[1], s->cells[_i].dim[2]);
        cflags[_i] = s->cells[_i].flags;
    };
    parallel_for(s->nr_cells, func_copy_cell_data);

    for(did = 0 ; did < nr_devices ; did++) {
        cuda_safe_call(hipSetDevice(e->devices[did]));

        cuda_safe_call(hipMalloc(&dummy[did], sizeof(float) * s->nr_cells * 3));
        cuda_safe_call(hipMemcpy(dummy[did], corig, sizeof(float) * s->nr_cells * 3, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_corig), &dummy[did], sizeof(float *), 0, hipMemcpyHostToDevice));
        
        cuda_safe_call(hipMalloc(&dummy[did], sizeof(float3) * s->nr_cells));
        cuda_safe_call(hipMemcpy(dummy[did], cdims, sizeof(float3) * s->nr_cells, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_cdims), &dummy[did], sizeof(float3 *), 0, hipMemcpyHostToDevice));
        
        cuda_safe_call(hipMalloc(&dummy[did], sizeof(unsigned int) * s->nr_cells));
        cuda_safe_call(hipMemcpy(dummy[did], cflags, sizeof(unsigned int) * s->nr_cells, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_cflags), &dummy[did], sizeof(unsigned int *), 0, hipMemcpyHostToDevice));
    }
    free(corig);
    free(cdims);
    free(cflags);
        
    /* Set the constant pointer to the null potential and other useful values. */
    dscale = ((float)SHRT_MAX) /(3.0 * sqrt(s->h[0]*s->h[0]*s->span[0]*s->span[0] + s->h[1]*s->h[1]*s->span[1]*s->span[1] + s->h[2]*s->h[2]*s->span[2]*s->span[2]));
    for(did = 0 ;did < nr_devices ; did++) {
        cuda_safe_call(hipSetDevice(e->devices[did]));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_dt), &dt, sizeof(float), 0, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_cutoff), &cutoff, sizeof(float), 0, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_cutoff2), &cutoff2, sizeof(float), 0, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_maxdist), &cutoff, sizeof(float), 0, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_maxtype), &(e->max_type), sizeof(int), 0, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_dscale), &dscale, sizeof(float), 0, hipMemcpyHostToDevice));
    }

    /* Allocate and fill the task list. */
    if((tasks_cuda = (struct task_cuda *)malloc(sizeof(struct task_cuda) * s->nr_tasks)) == NULL)
        return error(engine_err_malloc);
    if((cellsorts = (int *)malloc(sizeof(int) * s->nr_tasks)) == NULL)
        return error(engine_err_malloc);
    for(did = 0 ;did < nr_devices ; did++) {
        if((e->cells_cuda_local[did] = (int *)malloc(sizeof(int) * s->nr_cells)) == NULL)
            return error(engine_err_malloc);
        e->cells_cuda_nr[did]=0;
        cuda_safe_call(hipSetDevice(e->devices[did]));
        /* Select the tasks for each device ID. */  
        for(nr_tasks = 0, i = 0 ; i < s->nr_tasks ; i++) {
            
            /* Get local pointers. */
            t = &s->tasks[i];
            tc = &tasks_cuda[nr_tasks];
	    
            /* Skip pairs and self with wrong cid, keep all sorts. */
            if((t->type == task_type_pair && e->s.cells[t->i].GPUID != did  /*t->i % nr_devices != did */) ||
                (t->type == task_type_self && e->s.cells[t->i].GPUID != did /*e->s.cells[t->i].loc[1] < e->s.cdim[1] / e->nr_devices * (did + 1) && e->s.cells[t->i].loc[1] >= e->s.cdim[1] / e->nr_devices * did t->i % e->nr_devices != did*/))
                continue;
            
            /* Copy the data. */
            tc->type = t->type;
            tc->subtype = t->subtype;
            tc->wait = 0;
            tc->flags = t->flags;
            tc->i = t->i;
            tc->j = t->j;
            tc->nr_unlock = 0;
            
            /* Remember which task sorts which cell. */
            if(t->type == task_type_sort) {
                tc->flags = 0;
                cellsorts[ t->i ] = nr_tasks;
            }

            /*Add the cell to list of cells for this GPU if needed*/
            c1 = t->i >= 0; c2 = t->j >= 0;
            for(int i = 0; i < e->cells_cuda_nr[did] ; i++) {
                if(c1 == 0 && c2 == 0) 
                    break;
                /* Check cell is valid */
                if(t->i == e->cells_cuda_local[did][i])
                    c1 = 0;
                if(t->j == e->cells_cuda_local[did][i])
                    c2 = 0;
            }
            if(c1)
                e->cells_cuda_local[did][e->cells_cuda_nr[did]++] = t->i;
            if(c2)
                e->cells_cuda_local[did][e->cells_cuda_nr[did]++] = t->j;	                
            /* Add one task. */
            nr_tasks += 1;
		
        }

        /* Link each pair task to its sorts. */
        for(i = 0 ; i < nr_tasks ; i++) {
            tc = &tasks_cuda[i];
	
            if(tc->type == task_type_pair) {
                ts = &tasks_cuda[ cellsorts[ tc->i ] ];
                ts->flags |= (1 << tc->flags);
                ts->unlock[ ts->nr_unlock ] = i;
                ts->nr_unlock += 1;
                ts = &tasks_cuda[ cellsorts[ tc->j ] ];
                ts->flags |= (1 << tc->flags);
                ts->unlock[ ts->nr_unlock ] = i;
                ts->nr_unlock += 1;
            }
        }
        
        /* Set the waits. */
        for(i = 0 ; i < nr_tasks ; i++)
            for(k = 0 ; k < tasks_cuda[i].nr_unlock ; k++)
                tasks_cuda[ tasks_cuda[i].unlock[k] ].wait += 1;

        /* Allocate and fill the tasks list on the device. */
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_nr_tasks), &nr_tasks, sizeof(int), 0, hipMemcpyHostToDevice));
        cuda_safe_call(hipMalloc(&dummy[did], sizeof(struct task_cuda) * s->nr_tasks));
        cuda_safe_call(hipMemcpy(dummy[did], tasks_cuda, sizeof(struct task_cuda) * s->nr_tasks, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_tasks), &dummy[did], sizeof(struct task_cuda *), 0, hipMemcpyHostToDevice));
            
        /* Remember the number of tasks. */
        e->nrtasks_cuda[did] = nr_tasks;
            
    }
    
	/* Clean up */
    free(tasks_cuda);
    free(cellsorts);
    
    for(did = 0 ;did < nr_devices ; did++) {
        cuda_safe_call(hipSetDevice(e->devices[did]));

        /* Allocate the sortlists locally and on the device if needed. */
        cuda_safe_call(hipMalloc(&e->sortlists_cuda[did], sizeof(unsigned int) * s->nr_parts * 13));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_sortlists), &e->sortlists_cuda[did], sizeof(unsigned int *), 0, hipMemcpyHostToDevice));
        
	    /* Allocate the cell counts and offsets. */
        if((e->counts_cuda_local[did] = (int *)malloc(sizeof(int) * s->nr_cells)) == NULL ||
           (e->ind_cuda_local[did] = (int *)malloc(sizeof(int) * s->nr_cells)) == NULL)
            return error(engine_err_malloc);
        cuda_safe_call(hipMalloc(&e->counts_cuda[did], sizeof(int) * s->nr_cells));
        cuda_safe_call(hipMalloc(&e->ind_cuda[did], sizeof(int) * s->nr_cells));
        
        /* Allocate and init the taboo list on the device. */
        cuda_safe_call(hipMalloc(&dummy[did], sizeof(int) * s->nr_cells));
        cuda_safe_call(hipMemset(dummy[did], 0, sizeof(int) * s->nr_cells));
        cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_taboo), &dummy[did], sizeof(int *), 0, hipMemcpyHostToDevice));
    }

    // Allocate boundary conditions
    if(engine_cuda_boundary_conditions_load(e) < 0)
        return error(engine_err);
        
    if(engine_cuda_allocate_particles(e) < 0)
        return error(engine_err);

    if(engine_cuda_load_pots(e) < 0)
        return error(engine_err);

    if(engine_cuda_load_fluxes(e) < 0)
        return error(engine_err);
        
    if(engine_cuda_allocate_particle_states(e) < 0)
        return error(engine_err);
    
    /* Init the pair queue on the device. */
    if(engine_cuda_queues_load(e) < 0)
        return error(engine_err);
        
    /* He's done it! */
    return engine_err_ok;
    
}


/**
 * @brief Removes the potentials and cell pairs on the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
extern "C" int engine_parts_finalize(struct engine *e) {

    if(engine_cuda_boundary_conditions_finalize(e) < 0)
        return error(engine_err);

    if(cuda::engine_cuda_unload_pots(e) < 0)
        return error(engine_err);

    if(engine_cuda_finalize_particles(e) < 0)
        return error(engine_err);

    if(cuda::engine_cuda_finalize_particle_states(e) < 0)
        return error(engine_err);

    if(cuda::engine_cuda_unload_fluxes(e) < 0)
        return error(engine_err);

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call(hipSetDevice(e->devices[did]));

        e->nrtasks_cuda[did] = 0;

        // Free the sort list, counts and indices

        cuda_safe_call(hipFree(e->sortlists_cuda[did]));

        cuda_safe_call(hipFree(e->counts_cuda[did]));

        cuda_safe_call(hipFree(e->ind_cuda[did]));

    }

    return engine_err_ok;
}

/**
 * @brief Unload the potentials and cell pairs on the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
extern "C" int cuda::engine_cuda_finalize(struct engine *e) {
    if(engine_parts_finalize(e) < 0)
        return error(engine_err);

    if(engine_cuda_queues_finalize(e) < 0)
        return error(engine_err);

    return engine_err_ok;
}

/**
 * @brief Refresh the engine image on the CUDA device. 
 * 
 * Can be safely called while on the CUDA device to reload all data from the engine. 
 * 
 * @param e The #engine
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err)
 */
extern "C" int cuda::engine_cuda_refresh(struct engine *e) {
    
    if(engine_cuda_finalize(e) < 0)
        return error(engine_err);

    if(engine_cuda_load(e) < 0)
        return error(engine_err);

    for(int did = 0; did < e->nr_devices; did++) {

        cuda_safe_call(hipSetDevice(e->devices[did]));

        cuda_safe_call(hipDeviceSynchronize());

    }

    return engine_err_ok;
}
